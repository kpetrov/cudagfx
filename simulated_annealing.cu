#include "hip/hip_runtime.h"




__device__ int dev_get_a0(double *a0, double kb, float* rndgauss_field, float* rndunif_field, int ind){

  float a,b,y,help;
  
  a = abs(rndgauss_field[ind]);  //  need gaussian dist
  b = -log(1.0f - rndunif_field[4*ind]);   // we need 4 unif randoms per site , use the 0th here

  
  
  y = a*a + b;
  y = y/kb;
  
  help =  rndunif_field[4*ind+1];  // we need 4 unif randoms per site , use the 1st here
  
  if((2.0f * (help*help)) <= (2.0f -y)){
    (*a0) = (double)(1.0f - y);
    return(1);
  }
  else{
   (*a0) = 0.0;
   return(0);
  }
}





__device__ void dev_heatbath_su2(dev_su2* alpha, dev_su2* w, double beta, float* rndgauss_field, float* rndunif_field, int ind){
  
  dev_su2 v, a;
  double k, rdet, cos_theta, sin_theta, phi, norm;
  int ret;
  
  k =  (*w).a.x*(*w).a.x + (*w).a.y*(*w).a.y + (*w).b.x*(*w).b.x + (*w).b.y*(*w).b.y ;
  rdet = rsqrt(k);
  
  v.a.x = (*w).a.x*rdet;
  v.a.y = (*w).a.y*rdet;
  v.b.x = (*w).b.x*rdet;
  v.b.y = (*w).b.y*rdet;
  
  
  ret = dev_get_a0(&(a.a.x), k*beta, rndgauss_field, rndunif_field, ind);
  
    norm = sqrt(1.0 - a.a.x*a.a.x);
    cos_theta = 2.0 * rndunif_field[4*ind+2] - 1.0;  // we need 4 unif randoms per site , use the 2nd here 
    sin_theta = sqrt( 1.0 - cos_theta*cos_theta );
    phi = 6.2831853071795862 * (double) rndunif_field[4*ind+3]; // we need 4 unif randoms per site , use the 3rd here
  
    sincos(phi, &(a.b.x), &(a.a.y));
    
    a.a.y = norm * sin_theta * a.a.y;
    a.b.x = norm * sin_theta * a.b.x; 
    a.b.y = norm * cos_theta;
    
   if(ret == 1){  
   // a0 was accepted in the first place in dev_get_a0 
     dev_su2_ti_su2(alpha,&a,&v);
   }
   else{
   // a0 was not accepted  in the first place -> no update -> put alpha = w  
     (*alpha).a.x = (*w).a.x;
     (*alpha).a.y = (*w).a.y;
     (*alpha).b.x = (*w).b.x;
     (*alpha).b.y = (*w).b.y;
   }
  
}







__device__ void cabibbo_marinari_heatbath(dev_su3 * g, dev_su3 * star, double beta, float* rndgauss_field, float* rndunif_field, int ind){

// this routine does an relaxation update of an SU(3) matrix g


  int a,b,c;
  dev_su3 X;
  dev_su2 w, alpha;
  dev_complex dummy, dummy2, dummy3;

  
    for(a=0; a<2; a++){
      for(b=a+1; b<3; b++){
      
      dev_su3_ti_su3(&(X), g, star);
      
  
  w.a.x = X[a][a].re + X[b][b].re;
  w.b.y = -X[a][a].im + X[b][b].im;
  w.a.y = -X[a][b].im - X[b][a].im; 
  w.b.x = -X[a][b].re + X[b][a].re;
  
 
  
  /*
     my definitions: U = w0 ID + i ( w1 sigma1 + w2 sigma2 + w3 sigma3 )
     
               | 0  1 |           | 0  -i |            | 1  0 |
     sigma1 =  |      |  sigma2 = |       |   sigma3 = |      |
               | 1  0 |           | i   0 |            | 0 -1 |
  */
  
    
      dev_heatbath_su2(&(alpha), &(w), beta, rndgauss_field, rndunif_field,  ind);  
       
        
      for(c=0; c<3; c++){
      
        
      //dummy  =  cmplx(alpha(0),alpha(3), kind=RKIND) * u(a,c)     &
      //             + cmplx(alpha(2),alpha(1), kind=RKIND) * u(b,c)

       dummy = dev_cmult(dev_initcomplex(alpha.a.x,alpha.b.y),(*g)[a][c]);
       dummy2 = dev_cmult(dev_initcomplex(alpha.b.x,alpha.a.y),(*g)[b][c]);
       dummy = dev_cadd(dummy, dummy2);
       
       //     u(b,c) =  cmplx(-alpha(2), alpha(1), kind=RKIND) * u(a,c)   &
       //             + cmplx( alpha(0),-alpha(3), kind=RKIND) * u(b,c)
       
       dummy2 = dev_cmult(dev_initcomplex(-alpha.b.x,alpha.a.y),(*g)[a][c]);
       dummy3 = dev_cmult(dev_initcomplex(alpha.a.x,-alpha.b.y),(*g)[b][c]); 
       (*g)[b][c] = dev_cadd(dummy2, dummy3);
       
       //     u(a,c) = dummy   
       
       (*g)[a][c] = dummy;
       
      } 
         
    }
  } 
}







// do a trafo heatbath update of either the even or the odd sites depending on the dev_indeo_thissite and dev_indeo_nextside
// index fields

__global__ void dev_heatbath_sweep(dev_su3_2v * trafo_new, dev_su3_2v * gf, dev_su3_2v * trafo, int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn, float* rndgauss_field, float* rndunif_field){

    int eofieldpos, pos,hoppos,mu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 trafosmem[BLOCK];
    
    dev_su3 help, star;
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){
  
  pos = dev_indeo_thissite[eofieldpos];
  dev_su3zero( &(star) );
  
  // calculate the STAR
  
    //#pragma unroll 4
    for(mu=0;mu<4;mu++){
 //positive dir
      hoppos = dev_nn[8*pos+mu];
      //gauge_field  U_mu(x)
      #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+mu),&(gfsmem[ix]));
      #else
        dev_reconstructgf_2vtexref(gf, (4*pos+mu),&(gfsmem[ix]));
      #endif
      //trafo_field  g^+(x+mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif
      
      
      //dev_su3_ti_su3( &(gt), &(gfsmem[ix]), &(trafosmem[ix]) );    
      //dev_su3_add( &(star), &(gt));
      dev_add_su3_ti_su3(&(star) , &(gfsmem[ix]), &(trafosmem[ix]) );
      
  //negative dir
      hoppos = dev_nn[8*pos+4+mu];
      //gauge_field  U_mu(x-mu)^+
      #ifdef GF_8
       dev_reconstructgf_8texref_dagger(gf, 4*hoppos+mu,&(gfsmem[ix]));
      #else
       dev_reconstructgf_2vtexref_dagger(gf, 4*hoppos+mu,&(gfsmem[ix]));
      #endif
      
      //trafo_field  g^+(x-mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif 
      //dev_su3_ti_su3( &(gt), &(gfsmem[ix]), &(trafosmem[ix]) );
      //dev_su3_add( &(star), &(gt));
      dev_add_su3_ti_su3( &(star), &(gfsmem[ix]), &(trafosmem[ix]) );
  
   }

    //load g(x)
      #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
      #endif
    
    
   dev_su3copy( &(help) , &(trafosmem[ix]) );  
   cabibbo_marinari_heatbath( &(help),  &(star) , sa_beta, rndgauss_field, rndunif_field, eofieldpos); //also append the position of g, for the rng!
   								 //the global sa_beta is used for beta
   								 
   // now the old trafo field is in trafosmem the new field in help 
   dev_su3_normalize(&(help)); 
   
   #ifdef GF_8
     dev_storetrafo_8(pos, trafo_new ,&(help) );
   #else
     dev_storetrafo_2v(pos, trafo_new ,&(help) );
   #endif 
   
   
   #ifdef USETEXTURE
     // copy the trafofields of the sites that are not updated to destination field
     // e.g. if EVEN is updated just copy ODD trafos
     pos = dev_indeo_nextside[eofieldpos];
     //load g(x)
     #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
     #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
     #endif
     // and store it
     #ifdef GF_8
       dev_storetrafo_8(pos, trafo_new ,&(trafosmem[ix]) );
     #else
       dev_storetrafo_2v(pos, trafo_new ,&(trafosmem[ix]) );
     #endif    
   #endif

  }
}







void set_sa_temperature(int i){

    // Returns the temperature for a given i using a function 
    // proposed by Peter Schemel.
    double a, temperature, beta;
    
    //printf("Setting new SA temp: %d\n",i);
    if(saparam.Tmax == saparam.Tmin){
      temperature = saparam.Tmax;
    }   
    else{
       if(saparam.expo == 0){
          a = (double)(i) / (double) (saparam.N-1);
          temperature = pow( ( (double) saparam.Tmin/ (double) saparam.Tmax) , a) * (double) saparam.Tmax ;
       }
       else if(saparam.expo == -1){
          a = (double)(saparam.Tmin - saparam.Tmax) / (double) (saparam.N-1);    
          temperature = (a*i + saparam.Tmax);
       }
       else{
          a = pow(saparam.Tmin, -saparam.expo)- pow(saparam.Tmax, -saparam.expo);
          a = a / (double) (saparam.N-1);
          temperature = pow( (a*i + pow(saparam.Tmax,-saparam.expo))  , (-1.0/saparam.expo) );
       }
    }
   
   
   beta = 1.0/temperature; // -> beta = 1.0/(3 T)   , 3 for NCOL!!
   
   //printf("new SA temp = %f\n", temperature); 
   //printf("Tmin = %f, Tmax = %f, N = %d, expo = %f\n", saparam.Tmin, saparam.Tmax, saparam.N, saparam.expo);
   
   //set this beta on device
   CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL("sa_beta"), &beta, sizeof(double)) ) ;
}








// perform simulated annealing gauge fixing
void simannealing_gauge(){ 

  int gridsize;
  double maxdada = 0.0;
  int i;
  clock_t start, stop; 
  double timeelapsed = 0.0;
  hipError_t cudaerr;
  
  if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( (VOLUME/2) >= BLOCK){
   gridsize =VOLUME/2/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 

  
  
// Start timer
  assert((start = clock())!=-1);  
  
 #ifdef USETEXTURE
   bind_texture_gf(dev_gf);
 #endif


 for(i=0; i<saparam.N; i++){
   //set the temperature
   set_sa_temperature(i);
   hipDeviceSynchronize();
   cudaerr = hipGetLastError();
   if(cudaerr != hipSuccess){
     printf("%s\n", hipGetErrorString(cudaerr)); 
   } 
   // heatbath update
   
   //printf("Doing heatbath sweep...\n");
   hipGetLastError();
  #ifdef USETEXTURE
    // update of EVEN  
     bind_texture_trafo(dev_trafo1);
     dev_heatbath_sweep<<< griddim, blockdim >>> (dev_trafo2, dev_gf, dev_trafo1, 
                                                dev_eoidx_even, dev_eoidx_odd, 
                                                dev_nn, dev_rndgauss_field,
                                                 dev_rndunif_field); 
     unbind_texture_trafo();
     hipDeviceSynchronize();
     cudaerr = hipGetLastError();
     if(cudaerr != hipSuccess){
       printf("%s\n", hipGetErrorString(cudaerr)); 
     }
     
     
// generate new random numbers
    //printf("Updating the random numbers...\n");
    hipGetLastError();
    update_MT();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }     
     
   // update of ODD
     bind_texture_trafo(dev_trafo2);
           dev_heatbath_sweep<<< griddim, blockdim >>> (dev_trafo1, dev_gf,
                                                     dev_trafo2, dev_eoidx_odd,
                                                      dev_eoidx_even, dev_nn ,
                                                      dev_rndgauss_field, dev_rndunif_field);
     unbind_texture_trafo();
     hipDeviceSynchronize();
     cudaerr = hipGetLastError();
     if(cudaerr != hipSuccess){
       printf("%s\n", hipGetErrorString(cudaerr)); 
     }   
  
   // generate new random numbers
    //printf("Updating the random numbers...\n");
    hipGetLastError();
    update_MT();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }   
          
  #else  //USETEXTURE

    // update of EVEN  
       dev_heatbath_sweep<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, 
                                                  dev_eoidx_even, dev_eoidx_odd,
                                                   dev_nn, dev_rndgauss_field,
                                                    dev_rndunif_field); 
    hipDeviceSynchronize();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }


   // generate new random numbers
    //printf("Updating the random numbers...\n");
    hipGetLastError();
    update_MT();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }    
    
   // update of ODD
         dev_heatbath_sweep<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, 
                                                       dev_eoidx_odd, dev_eoidx_even,
                                                      dev_nn, dev_rndgauss_field, 
                                                      dev_rndunif_field);  
    hipDeviceSynchronize();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }
  
   // generate new random numbers
    //printf("Updating the random numbers...\n");
    hipGetLastError();
    update_MT();
    cudaerr = hipGetLastError();
    if(cudaerr != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr)); 
    }      
  #endif  //USETEXTURE
    
    
    

    if((i%saparam.checkint) == 0){    
      //printf("recalculating functional...\n");
   //recalculate dAdA
    #ifdef USETEXTURE
       bind_texture_trafo(dev_trafo1);
    #endif
      hipGetLastError();
      maxdada = calc_functional(dev_gf, dev_trafo1);   
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }
    #ifdef USETEXTURE
      unbind_texture_trafo();
    #endif      
      
      printf("iter %6d:\t FUNC = %.16e \t dAdA = %.16e\t max(dAdA) = %.16e\n",i, FUNC, DADA, maxdada);
    }
   

  }//i
 
  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif 

  assert((stop = clock())!=-1);
  timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
  printf("SA finished after %f sec\n", timeelapsed);

}























