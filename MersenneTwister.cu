#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/*
 * This sample implements Mersenne Twister random number generator 
 * and Cartesian Box-Muller transformation on the GPU.
 * See supplied whitepaper for more explanations.
 */


#include "MersenneTwister.h"



///////////////////////////////////////////////////////////////////////////////
// Common host and device function 
///////////////////////////////////////////////////////////////////////////////
//ceil(a / b)
extern "C" int iDivUp(int a, int b){
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

//floor(a / b)
extern "C" int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
extern "C" int iAlignUp(int a, int b){
    return ((a % b) != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
extern "C" int iAlignDown(int a, int b){
    return a - a % b;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////
static int    PATH_N_GAUSS;
static int    PATH_N_UNIF;
static int    N_PER_RNG_GAUSS;
static int    N_PER_RNG_UNIF;
static int    RAND_N_GAUSS;
static int    RAND_N_UNIF;





__device__  mt_struct_stripped ds_MT[MT_RNG_COUNT];
static mt_struct_stripped h_MT[MT_RNG_COUNT];
__device__ unsigned int d_mtstatus[MT_RNG_COUNT][MT_NN];


// fields for the MT random number generator
//__device__ float * dev_rndunif_field;
//__device__ float * dev_rndgauss_field;




//Load twister configurations
void loadMTGPU(const char *fname){
    FILE *fd = fopen(fname, "rb");
    if(!fd){
        printf("initMTGPU(): failed to open %s\n", fname);
        printf("FAILED\n");
        exit(0);
    }
    if( !fread(h_MT, sizeof(h_MT), 1, fd) ){
        printf("initMTGPU(): failed to load %s\n", fname);
        printf("FAILED\n");
        exit(0);
    }
    fclose(fd);
}




//Initialize/seed twister for current GPU context
void seedMTGPU(){
    int i;
    //Need to be thread-safe
    mt_struct_stripped *MT = (mt_struct_stripped *)malloc(MT_RNG_COUNT * sizeof(mt_struct_stripped));

    /* initialize poor rng: */
    srand ( time(NULL) );

    /* initialize MT rng seeds */
    for(i = 0; i < MT_RNG_COUNT; i++){
        MT[i]      = h_MT[i];
        MT[i].seed = (unsigned int) rand();
    }
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ds_MT), MT, sizeof(h_MT)) );

    free(MT);
}



//Save twister for current GPU context
void saveMTGPU(const char *fname){
    FILE *fd = fopen(fname, "w");
    if(!fd){
        printf("saveMTGPU(): failed to open %s\n", fname);
        printf("FAILED\n");
        exit(0);
    }
    fwrite(h_MT, sizeof(h_MT), 1, fd);
    fclose(fd);
}




////////////////////////////////////////////////////////////////////////////////
// Write MT_RNG_COUNT vertical lanes of NPerRng random numbers to *d_Random.
// For coalesced global writes MT_RNG_COUNT should be a multiple of warp size.
// Initial states for each generator are the same, since the states are
// initialized from the global seed. In order to improve distribution properties
// on small NPerRng supply dedicated (local) seed to each twister.
// The local seeds, in their turn, can be extracted from global seed
// by means of any simple random number generator, like LCG.
////////////////////////////////////////////////////////////////////////////////
__global__ void RandomGPU(
    float *d_Random,
    int NPerRng, int initialized
){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    int iState, iState1, iStateM, iOut;
    unsigned int mti, mti1, mtiM, x;
    unsigned int mt[MT_NN];

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N){
        //Load bit-vector Mersenne Twister parameters
        mt_struct_stripped config = ds_MT[iRng];

       if(!initialized){
         // initialize seed and construct status mt must be initialized from host before
         mt[0] = ds_MT[iRng].seed;
         for(iState = 1; iState < MT_NN; iState++)
            mt[iState] = (1812433253U * (mt[iState - 1] ^ (mt[iState - 1] >> 30)) + iState) & MT_WMASK;
       } 
       else{
         for(iState = 0; iState < MT_NN; iState++) mt[iState] = d_mtstatus[iRng][iState];
       } 
        
        iState = 0;
        mti1 = mt[0];
        for(iOut = 0; iOut < NPerRng; iOut++){
            //iState1 = (iState +     1) % MT_NN
            //iStateM = (iState + MT_MM) % MT_NN
            iState1 = iState + 1;
            iStateM = iState + MT_MM;
            if(iState1 >= MT_NN) iState1 -= MT_NN;
            if(iStateM >= MT_NN) iStateM -= MT_NN;
            mti  = mti1;
            mti1 = mt[iState1];
            mtiM = mt[iStateM];

            x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
            x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? config.matrix_a : 0);
            mt[iState] = x;
            iState = iState1;

            //Tempering transformation
            x ^= (x >> MT_SHIFT0);
            x ^= (x << MT_SHIFTB) & config.mask_b;
            x ^= (x << MT_SHIFTC) & config.mask_c;
            x ^= (x >> MT_SHIFT1);

            //Convert to (0, 1] float and write to global memory
            d_Random[iRng + iOut * MT_RNG_COUNT] = ((float)x + 1.0f) / 4294967296.0f;
        }
        
       // save status of mt
       ds_MT[iRng].seed = mt[0]; 
       for(iState = 0; iState < MT_NN; iState++) d_mtstatus[iRng][iState] = mt[iState]; 
    }
}



////////////////////////////////////////////////////////////////////////////////
// Transform each of MT_RNG_COUNT lanes of NPerRng uniformly distributed 
// random samples, produced by RandomGPU(), to normally distributed lanes
// using Cartesian form of Box-Muller transformation.
// NPerRng must be even.
////////////////////////////////////////////////////////////////////////////////
#define PIf 3.14159265358979f
__device__ inline void BoxMuller(float& u1, float& u2){
    float   r = sqrtf(-2.0f * logf(u1));
    float phi = 2 * PIf * u2;
    u1 = r * __cosf(phi);
    u2 = r * __sinf(phi);
}



__global__ void BoxMullerGPU(float *d_Random, int NPerRng){
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    for(int iRng = tid; iRng < MT_RNG_COUNT; iRng += THREAD_N)
        for(int iOut = 0; iOut < NPerRng; iOut += 2)
            BoxMuller(
                d_Random[iRng + (iOut + 0) * MT_RNG_COUNT],
                d_Random[iRng + (iOut + 1) * MT_RNG_COUNT]
            );
}






extern "C" void init_MT(int n_gaussnumbers, int n_unifnumbers){
 
 
  hipError_t cudaerr;
  //determine sizes for gauss numbers
  
  printf("Initializing MT random number generator...\n");
  
  
  PATH_N_GAUSS = n_gaussnumbers;
  N_PER_RNG_GAUSS = iAlignUp(iDivUp(PATH_N_GAUSS, MT_RNG_COUNT), 2);
  RAND_N_GAUSS = MT_RNG_COUNT * N_PER_RNG_GAUSS;
  printf("No. of gauss random numbers: %d\n", RAND_N_GAUSS );
   
  //determine sizes for unif. numbers 
  PATH_N_UNIF = n_unifnumbers;
  N_PER_RNG_UNIF = iAlignUp(iDivUp(PATH_N_UNIF, MT_RNG_COUNT), 2);
  RAND_N_UNIF = MT_RNG_COUNT * N_PER_RNG_UNIF; 
  printf("No. of unif. dist. random numbers: %d\n", RAND_N_UNIF );
   
  // load and initialize twister configurations on device 
  // seed the twisters
  const char *dat_path = "MersenneTwister.dat";
  printf("Loading GPU twisters configurations from file %s...\n", dat_path);
  loadMTGPU(dat_path);
  seedMTGPU();

  //allocate fields for random numbers
  printf("Allocating device memory for random numbers...\n");
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_rndgauss_field, RAND_N_GAUSS * sizeof(float)) );
  CUDA_SAFE_CALL(hipMalloc((void **)&dev_rndunif_field, RAND_N_UNIF * sizeof(float)));


 // CREATE FIRST RANDOM NUMBERS 
 /* update the random field for gauss numbers -> BoxMuller afterwards*/
    hipDeviceSynchronize();
    RandomGPU<<<32, 128>>>(dev_rndgauss_field, N_PER_RNG_GAUSS,0);
    BoxMullerGPU<<<32, 128>>>(dev_rndgauss_field, N_PER_RNG_GAUSS);

  /* update the random field for unif. dist. numbers*/
    hipDeviceSynchronize();
    RandomGPU<<<32, 128>>>(dev_rndunif_field, N_PER_RNG_UNIF,0);
    hipDeviceSynchronize();
   


   cudaerr = hipGetLastError();
   if(cudaerr != hipSuccess){
     printf("%s\n", hipGetErrorString(cudaerr)); 
   } 



}





extern "C" void update_MT(){


  /* update the random field for gauss numbers -> BoxMuller afterwards*/
    hipDeviceSynchronize();
    RandomGPU<<<32, 128>>>(dev_rndgauss_field, N_PER_RNG_GAUSS,1);
    BoxMullerGPU<<<32, 128>>>(dev_rndgauss_field, N_PER_RNG_GAUSS);
    hipDeviceSynchronize();

    
  /* update the random field for unif. dist. numbers*/  
    RandomGPU<<<32, 128>>>(dev_rndunif_field, N_PER_RNG_UNIF, 1);
    hipDeviceSynchronize();
    
/*
   float * blub = (float*) malloc(RAND_N_UNIF*sizeof(float));
   printf("%d \n", RAND_N_UNIF);
   
   CUDA_SAFE_CALL(hipMemcpy(blub, dev_rndunif_field, (size_t)(RAND_N_UNIF*sizeof(float)), hipMemcpyDeviceToHost));
   for(int k=0; k<4; k++){
     for(int j=VOLUME/2-10; j<VOLUME/2; j++){
       printf("%f, ", blub[4*j+k]);
     }
   }
   printf("\n\n");
   free(blub);
 */

}



extern "C" void finalize_MT(){
  hipFree(dev_rndgauss_field);
  hipFree(dev_rndunif_field);
}





