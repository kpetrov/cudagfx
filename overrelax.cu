#include "hip/hip_runtime.h"






// init the gpu inner solver, assign constants etc.
__global__ void dev_gfix_init (int* grid){
  dev_LX = grid[0];
  dev_LY = grid[1];
  dev_LZ = grid[2];
  dev_T = grid[3];
  dev_VOLUME = grid[4]; // grid[4] is initialized 1/2 VOLUME for eo
}


// init the gpu inner solver, assign constants etc.
__global__ void dev_check_geom (int* grid){
  grid[0] = dev_LX;
  grid[1] = dev_LY;
  grid[2] = dev_LZ;
  grid[3] = dev_T;
  grid[4] = dev_VOLUME;
}




// calculates the mean plaquette of the gauge field
// uses 2d parallelization:
// N_grid = T, N_block = LZ
// CPU has to do last summation over T
// BLOCKPLAQ MUST be larger than LZ !!! -> define correctly in cudaglobal.h
__global__ void dev_mean_plaq(double* reductionfield_plaq, int * dev_nn, dev_su3_2v * gf){
  double mplaq = 0.0;
  int x0pos, x1pos, x2pos ; /* x0pos = basepoint of plaquette, x1pos = x0pos + e_mu, x2pos = x0pos + e_nu */
  int iz,x,y,z,t,mu,nu;
  dev_su3 su3matrix,su3matrix2, M1,M2,M3,M4;
  dev_complex chelp;
  


  
  __shared__ double output[BLOCKPLAQ];
  t = blockIdx.x;
  z = threadIdx.x;
  

      for(y=0; y<dev_LY; y++){
        for(x=0; x<dev_LX; x++){
          for(nu=0;nu <3; nu++){
            for(mu =nu+1; mu < 4; mu++){
              x0pos = x + dev_LX*(y + dev_LY*(z + dev_LZ*t));
              x1pos = dev_nn[8*x0pos + mu];
              x2pos = dev_nn[8*x0pos + nu];          

/* U_nu(x) */
            #ifdef GF_8
              dev_reconstructgf_8texref(gf, (4*x0pos+mu),&M1);
            #else
              dev_reconstructgf_2vtexref(gf, (4*x0pos+mu),&M1);
            #endif
/* U_mu(x+e_mu) */
            #ifdef GF_8
              dev_reconstructgf_8texref(gf, (4*x1pos+nu),&M2);
            #else
              dev_reconstructgf_2vtexref(gf, (4*x1pos+nu),&M2);
            #endif

/* Udagger_nu(x+e_nu) */
            #ifdef GF_8
              dev_reconstructgf_8texref_dagger(gf, (4*x2pos+mu),&M3);
            #else
              dev_reconstructgf_2vtexref_dagger(gf, (4*x2pos+mu),&M3);
            #endif
/* Udagger_mu(x)*/
            #ifdef GF_8
              dev_reconstructgf_8texref_dagger(gf, (4*x0pos+nu),&M4);
            #else
              dev_reconstructgf_2vtexref_dagger(gf, (4*x0pos+nu),&M4);
            #endif
  
              /* multiply these and store in su3matrix*/
              dev_su3_ti_su3(&su3matrix, &M3,&M4);
              dev_su3_ti_su3(&su3matrix2, &M2,&su3matrix);
              dev_su3_ti_su3(&su3matrix, &M1,&su3matrix2);

              chelp = dev_su3trace(&su3matrix);
              mplaq += chelp.re/3.0; /* Realteile von Tr UUUU aufsummieren*/
            }
          }
           
        }
      } 
    output[z] = mplaq;

  __syncthreads();
  
  if(threadIdx.x == 0){
    
    /* normieren */
    double accum = 0.0;
    for(iz=0; iz < dev_LZ; iz++){
      accum += output[iz];  
    }
    accum = accum*(1.0/(6.0*dev_VOLUME));
    reductionfield_plaq[t] = accum;
  }
  __syncthreads();
  
}



double calc_plaquette(dev_su3_2v * U, int outputyn){
   double erg=0.0;
   int j;
   
   #ifdef USETEXTURE
    bind_texture_gf(U);
   #endif
   dev_mean_plaq <<< T , LZ >>> (dev_redfield_plaq, dev_nn, U) ;
   #ifdef USETEXTURE
   unbind_texture_gf();
   #endif   
   

   
   hipMemcpy(redfield_plaq, dev_redfield_plaq, (size_t)(T*sizeof(double)), hipMemcpyDeviceToHost);


   for(j=0; j<T; j++){
     erg+=redfield_plaq[j];
     //printf("%e\n", redfield_plaq[j]);
   }
   if(outputyn==1) printf("PLAQ = %.16f\n",erg);
   return(erg);
}






// Returns g = R^w * g  where R^w = SUM_{i=0..N} C(w,i)*{R}^i
// and R = gn * g^+ - 1 . Here C(w,i) = GAMMA(w+1)/ GAMMA(w+1-i)i!
// where GAMMA(n) is the gamma function 

__device__ void overrelax ( dev_su3 * g, dev_su3 * gn, double w, int N){

  int i;
  dev_su3 one, R, a, Rw, help;
  
  dev_unitsu3(&(one));
  
  dev_su3_ti_su3d (&(R), gn, g ); 
  dev_su3_sub(&(R), &(one)); // R=gn*g^+ - 1

  dev_su3_assign(&(Rw), &(one));
  dev_su3_assign(&(a), &(one));
  
  for(i=1; i<N; i++){
    // a = a*R
    dev_su3_ti_su3(&(help), &(a), &(R));
    dev_su3_assign(&(a), &(help));
    
    dev_su3_real_mult(&(a),  (1.0 + w - i)/i );
    dev_su3_add(&(Rw), &(a));
    
  }
  dev_su3_normalize(&(Rw));
  
  // g= Rw*g
  dev_su3_ti_su3(&(help), &(Rw), g);
  dev_su3_assign(g, &(help));
  
}






/* this is the SU(2) relax routine 
   the SU(2) matrices are double2s that represent the matrices: 
   u = double2.a.x * ID + double2.a.y * sigma1 + double2.b.x * sigma2 + double2.b.y * sigma3
*/
__device__ void relax_su2(dev_su2 * out, dev_su2 * in){
  double det = (*in).a.x*(*in).a.x + (*in).a.y*(*in).a.y + (*in).b.x*(*in).b.x + (*in).b.y*(*in).b.y;
  det = rsqrt(det); // 1/sqrt(det)
  
  
  (*out).a.x = (*in).a.x*det;  
  (*out).a.y = (*in).a.y*det;
  (*out).b.x = (*in).b.x*det;
  (*out).b.y = (*in).b.y*det;
 
}




__device__ void cabibbo_marinari_relax(dev_su3 * g, dev_su3 * star){

// this routine does an relaxation update of an SU(3) matrix g


  int a,b,c;
  dev_su3 X;
  dev_su2 w, alpha;
  dev_complex dummy, dummy2, dummy3;

  
    for(a=0; a<2; a++){
      for(b=a+1; b<3; b++){
      
      dev_su3_ti_su3(&(X), g, star);
      
  
  w.a.x = X[a][a].re + X[b][b].re;
  w.b.y = -X[a][a].im + X[b][b].im;
  w.a.y = -X[a][b].im - X[b][a].im; 
  w.b.x = -X[a][b].re + X[b][a].re;
  
 
  
  /*
     my definitions: U = w0 ID + i ( w1 sigma1 + w2 sigma2 + w3 sigma3 )
     
               | 0  1 |           | 0  -i |            | 1  0 |
     sigma1 =  |      |  sigma2 = |       |   sigma3 = |      |
               | 1  0 |           | i   0 |            | 0 -1 |
  */
  
    
      relax_su2(&(alpha), &(w));  
       
        
 
        
        
        
        
      for(c=0; c<3; c++){
      
        
      //dummy  =  cmplx(alpha(0),alpha(3), kind=RKIND) * u(a,c)     &
      //             + cmplx(alpha(2),alpha(1), kind=RKIND) * u(b,c)

       dummy = dev_cmult(dev_initcomplex(alpha.a.x,alpha.b.y),(*g)[a][c]);
       dummy2 = dev_cmult(dev_initcomplex(alpha.b.x,alpha.a.y),(*g)[b][c]);
       dummy = dev_cadd(dummy, dummy2);
       
       //     u(b,c) =  cmplx(-alpha(2), alpha(1), kind=RKIND) * u(a,c)   &
       //             + cmplx( alpha(0),-alpha(3), kind=RKIND) * u(b,c)
       
       dummy2 = dev_cmult(dev_initcomplex(-alpha.b.x,alpha.a.y),(*g)[a][c]);
       dummy3 = dev_cmult(dev_initcomplex(alpha.a.x,-alpha.b.y),(*g)[b][c]); 
       (*g)[b][c] = dev_cadd(dummy2, dummy3);
       
       //     u(a,c) = dummy   
       
       (*g)[a][c] = dummy;
       
      } 
         
    }
  } 
}



__global__ void dev_apply_trafo(dev_su3_2v * gf_new, dev_su3_2v * gf, dev_su3_2v * trafo, int* dev_nn){
int pos,hoppos,mu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 trafosmem[BLOCK];
     dev_su3  help;
    

  pos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(pos < dev_VOLUME){
  

    for(mu=0;mu<4;mu++){
      
      hoppos = dev_nn[8*pos+mu];
      //gauge_field  U_mu(x)
      #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+mu),&(gfsmem[ix]));
      #else
        dev_reconstructgf_2vtexref(gf, (4*pos+mu),&(gfsmem[ix]));
      #endif
      
      
      
      //trafo_field  g^+(x+mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif
      
      
      dev_su3_ti_su3(&(help), &(gfsmem[ix]),  &(trafosmem[ix]) );
      
      
      //trafo_field  g(x)
      #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
      #endif

      
      dev_su3_ti_su3(&(gfsmem[ix]), &(trafosmem[ix]), &(help) );
      
      
      
      #ifdef GF_8
       dev_storegf_8((4*pos+mu), gf_new ,&(gfsmem[ix]) );
      #else
          dev_storegf_2v((4*pos+mu), gf_new ,&(gfsmem[ix]) );
      #endif       
       
    }
  }
}



__global__ void dev_overrelax_step(dev_su3_2v * trafo_new, dev_su3_2v * gf, dev_su3_2v * trafo, int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn){
    int eofieldpos, pos,hoppos,mu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 trafosmem[BLOCK];
    
    dev_su3 help, star;
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){
  
  pos = dev_indeo_thissite[eofieldpos];
  dev_su3zero( &(star) );
  
  // calculate the STAR
    //#pragma unroll 4
    for(mu=0;mu<4;mu++){
 //positive dir
      hoppos = dev_nn[8*pos+mu];
      //gauge_field  U_mu(x)
      #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+mu),&(gfsmem[ix]));
      #else
        dev_reconstructgf_2vtexref(gf, (4*pos+mu),&(gfsmem[ix]));
      #endif
      //trafo_field  g^+(x+mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif
      
      
      //dev_su3_ti_su3( &(gt), &(gfsmem[ix]), &(trafosmem[ix]) );    
      //dev_su3_add( &(star), &(gt));
      dev_add_su3_ti_su3(&(star) , &(gfsmem[ix]), &(trafosmem[ix]) );
      
  //negative dir
      hoppos = dev_nn[8*pos+4+mu];
      //gauge_field  U_mu(x-mu)^+
      #ifdef GF_8
       dev_reconstructgf_8texref_dagger(gf, 4*hoppos+mu,&(gfsmem[ix]));
      #else
       dev_reconstructgf_2vtexref_dagger(gf, 4*hoppos+mu,&(gfsmem[ix]));
      #endif
      
      //trafo_field  g^+(x-mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif 
      //dev_su3_ti_su3( &(gt), &(gfsmem[ix]), &(trafosmem[ix]) );
      //dev_su3_add( &(star), &(gt));
      dev_add_su3_ti_su3( &(star), &(gfsmem[ix]), &(trafosmem[ix]) );
  
  }

    //load g(x)
      #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
      #endif
    
    
   /*
   dev_su3dagger(&(help),&(star));
   dev_su3copy( &(star),  &(help) ); 
   */
   
   dev_su3copy( &(help) , &(trafosmem[ix]) );
   
   cabibbo_marinari_relax( &(help),  &(star) );
   // now the old trafo field is in trafosmem the new field in help 
   dev_su3_normalize(&(help)); 
   

   overrelax(&(trafosmem[ix]), &(help), 1.68, 3);    
   dev_su3_normalize( &(trafosmem[ix]) ); 
   
   #ifdef GF_8
     dev_storetrafo_8(pos, trafo_new ,&(trafosmem[ix]) );
   #else
     dev_storetrafo_2v(pos, trafo_new ,&(trafosmem[ix]) );
   #endif 


   
   #ifdef USETEXTURE
     // copy the trafofields of the sites that are not updated to destination field
     // e.g. if EVEN is updated just copy ODD trafos
     pos = dev_indeo_nextside[eofieldpos];
     //load g(x)
     #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
     #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
     #endif
     // and store it
     #ifdef GF_8
       dev_storetrafo_8(pos, trafo_new ,&(trafosmem[ix]) );
     #else
       dev_storetrafo_2v(pos, trafo_new ,&(trafosmem[ix]) );
     #endif    
   #endif
   
  }//pos < dev_VOLUME
}




__device__ void dev_vectorpotential(dev_su3* U){
 
 dev_su3 temp;
 dev_complex i_half = dev_initcomplex(0.0, 0.5);
 dev_complex trace;
 
 dev_su3dagger(&(temp), U);
 dev_su3_sub( &(temp), U);  // U+ - U
 dev_su3skalarmult(U, i_half, &(temp)); // U -> - i/2 * (U - U^+)
 
 // get traceless part
 
 //trace
 trace = dev_su3trace(U);
 // trace/3
 trace.re = trace.re/3.0;
 trace.im = trace.im/3.0;
 // subtract from diagonal
 (*U)[0][0] = dev_csub((*U)[0][0], trace);
 (*U)[1][1] = dev_csub((*U)[1][1], trace);
 (*U)[2][2] = dev_csub((*U)[2][2], trace);
 
}






// calculates F = \sum_{mu,x} g(x) u_mu(x) g^+(x+mu)
// i.e. the functional for Landau gauge
__global__ void dev_functional(dev_su3_2v * gf, dev_su3_2v * trafo, int * dev_nn, double * reduction_field_F, double * reduction_field_dAdA, double * reduction_field_maxdAdA){
  
  int pos, mu, hoppos, i;

  __shared__ dev_su3 gfsmem[BLOCK];
  __shared__ dev_su3 trafosmem[BLOCK];
  __shared__ double Flocal[BLOCK];
  __shared__ double dAdAlocal[BLOCK];
  
  dev_su3 gather, help, fw, bw;
  
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  
  int ix = threadIdx.x;
  
  
  if(pos < dev_VOLUME){

   dev_su3zero(&(gather));

    Flocal[ix] = 0.0;
    dAdAlocal[ix] = 0.0;
    //#pragma unroll 4
    for(mu=0;mu<4;mu++){

//FORWARD      
      hoppos = dev_nn[8*pos+mu];
      
      //gauge_field x
      #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+mu),&(gfsmem[ix]));
      #else
        dev_reconstructgf_2vtexref(gf, (4*pos+mu),&(gfsmem[ix]));
      #endif
      
      
      //trafo field x
      #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, pos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref(trafo, pos,&(trafosmem[ix]));
      #endif         
      
      // help = g(x) * u(x) 
      dev_su3_ti_su3(&(help), &(trafosmem[ix]), &(gfsmem[ix]) );
      
      //g^+(x+mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, hoppos,&(trafosmem[ix]));
      #endif    
     
     // g u g+
      dev_su3_ti_su3(&(fw), &(help), &(trafosmem[ix]));
      Flocal[ix] += dev_su3Retrace(&(fw));


// BACKWARD
    hoppos = dev_nn[8*pos+4+mu];
      
      //gauge_field u(x-mu)
      #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*hoppos+mu),&(gfsmem[ix]));
      #else
        dev_reconstructgf_2vtexref(gf, (4*hoppos+mu),&(gfsmem[ix]));
      #endif
      
      
      //trafo field g(x-mu)
      #ifdef GF_8
        dev_reconstructtrafo_8texref(trafo, hoppos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref(trafo, hoppos,&(trafosmem[ix]));
      #endif         
      
      // help = g(x-mu) * u(x-mu) 
      dev_su3_ti_su3(&(help), &(trafosmem[ix]) , &(gfsmem[ix]));
      
      //g^+(x)
      #ifdef GF_8
        dev_reconstructtrafo_8texref_dagger(trafo, pos,&(trafosmem[ix]));
      #else
        dev_reconstructtrafo_2vtexref_dagger(trafo, pos,&(trafosmem[ix]));
      #endif    
   
   // g u g
   // backward term into bw
   dev_su3_ti_su3( &(bw), &(help), &(trafosmem[ix]) );
   
   dev_vectorpotential(&(fw));
   dev_vectorpotential(&(bw));
   dev_su3_sub(&(bw), &(fw)); // backward - forward
   
   
 // add up all backward - forward terms in gatherb  
  dev_su3_add(&(gather), &(bw));
 
   }//mu
  
  // sum_mu (dA) -> gather 
   // help = gather^+
   dev_su3_ti_su3d( &(help),&(gather), &(gather));  
   dAdAlocal[ix] += dev_su3Retrace( &(help)  ); 

  }// pos < dev_VOLUME
  
  
  __syncthreads();
  if(ix==0){
    reduction_field_F[blockIdx.x] = 0.0;
    reduction_field_dAdA[blockIdx.x] = 0.0;
    double actualmax = 0.0;
    for(i=0; i<blockDim.x; i++){
      reduction_field_F[blockIdx.x] += Flocal[i];
      reduction_field_dAdA[blockIdx.x] += dAdAlocal[i];
      if(dAdAlocal[i] > actualmax){
        actualmax = dAdAlocal[i];
      }
    }
    reduction_field_maxdAdA[blockIdx.x] = actualmax;
  }
  __syncthreads();
}




double calc_functional(dev_su3_2v * gf, dev_su3_2v * trafo){
  int i,gridsize;
  double F = 0.0;
  double dada = 0.0;
  double maxdada = 0.0;
  if(VOLUME%BLOCK != 0){
   printf("Error: VOLUME is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( VOLUME >= BLOCK){
   gridsize =VOLUME/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 
  
  
  int redfieldsize = VOLUME/BLOCK; 
   
   #ifdef USETEXTURE
     bind_texture_trafo(trafo);
   #endif
     dev_functional<<< griddim, blockdim >>> (gf, trafo, dev_nn, dev_redfield_F, dev_redfield_dAdA, dev_redfield_maxdAdA);
   #ifdef USETEXTURE
     unbind_texture_trafo();
   #endif
   
   
   CUDA_SAFE_CALL(hipMemcpy(redfield_F, dev_redfield_F, (size_t)(redfieldsize*sizeof(double)), hipMemcpyDeviceToHost));
   CUDA_SAFE_CALL(hipMemcpy(redfield_dAdA, dev_redfield_dAdA, (size_t)(redfieldsize*sizeof(double)), hipMemcpyDeviceToHost));
   CUDA_SAFE_CALL(hipMemcpy(redfield_maxdAdA, dev_redfield_maxdAdA, (size_t)(redfieldsize*sizeof(double)), hipMemcpyDeviceToHost));   
   
   for(i=0;i<redfieldsize;i++){
     // sum up F and dAdA
     F += redfield_F[i];
     dada += redfield_dAdA[i];
     // find maxdAdA
     if(redfield_maxdAdA[i] > maxdada){
       maxdada = redfield_maxdAdA[i];
     }
     
     /*
     if(i<redfieldsize){ 
      printf("%d  %f\n", i, redfield_F[i]);
     }
     */
   }
   
   F=F/(4.0*VOLUME);
   dada=dada/(VOLUME);
   
   // set global values  ///////
   FUNC=F;
   DADA=dada;
   maxDADA=maxdada;
   //////////////////////////
  

   return(maxdada);
}




// perform overrelaxation gauge fixing
int overrelax_gauge(int maxit, double eps, int checkinterval){ 

  int gridsize;
  double maxdada = 0.0;
  int i;
  clock_t start, stop; 
  double timeelapsed = 0.0;
  hipError_t cudaerr;
  
  
  if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( (VOLUME/2) >= BLOCK){
   gridsize =VOLUME/2/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 

  
  
// Start timer
  assert((start = clock())!=-1);  
  
 #ifdef USETEXTURE
   bind_texture_gf(dev_gf);
 #endif


  for(i=0; i<maxit; i++){


   // overrelax update
   #ifdef USETEXTURE
    // update of EVEN  
     bind_texture_trafo(dev_trafo1);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo2, dev_gf, dev_trafo1, dev_eoidx_even, dev_eoidx_odd, dev_nn); 
     unbind_texture_trafo();

   // update of ODD
     bind_texture_trafo(dev_trafo2);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo2, dev_eoidx_odd, dev_eoidx_even, dev_nn);
     unbind_texture_trafo();
   
    #else  //USETEXTURE

    // update of EVEN  
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_even, dev_eoidx_odd, dev_nn); 

   // update of ODD
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_odd, dev_eoidx_even, dev_nn);  
    #endif  //USETEXTURE
    
    
// test 64 bit error
 
//  int grid[5];
//  dev_check_geom<<< 1, 1 >>> (dev_grid);
//  /*
//  if((cudaerr=hipGetLastError()) != hipSuccess){
//       printf("%s\n", hipGetErrorString(cudaerr));
//       exit(200);
//     }
//   */ 
//  size_t dev_size = (size_t)  (5 * sizeof(int));
//  printf("size=%d\n", dev_size);
//  hipMemcpy(&(grid[0]), dev_grid, dev_size, hipMemcpyDeviceToHost);
//  printf("LX=%d, LY=%d, LZ=%d, T=%d, VOL=%d\n", grid[0], grid[1], grid[2], grid[3], grid[4]);
//  
//  
//  #ifdef GF_8
//    size_t dev_gfsize = 4*VOLUME * sizeof(dev_su3_8);
//    hipMemcpy(h2d_trafo, dev_trafo1, dev_gfsize, hipMemcpyDeviceToHost);
//    from8tosu3_trafo(trafo1, h2d_trafo);
//    show_su3(&(trafo1[10])); 
//   #else
//     size_t dev_gfsize = 6*VOLUME * sizeof(dev_su3_2v);
//     hipMemcpy(h2d_trafo, dev_trafo1, dev_gfsize, hipMemcpyDeviceToHost);
//     from2vtosu3_trafo(trafo1, h2d_trafo);
//     show_su3(&(trafo1[10]));
//   #endif

// test 64 bit error    
/*    
    
    
   if((cudaerr=hipGetLastError()) != hipSuccess){
      printf("%s\n", hipGetErrorString(cudaerr));
      exit(200);
    }*/




    if((i%checkinterval) == 0){    
      #ifdef USETEXTURE
        bind_texture_trafo(dev_trafo1);
      #endif
   //recalculate dAdA
      maxdada = calc_functional(dev_gf, dev_trafo1);   
      
      printf("iter %6d:\t FUNC = %.16e \t dAdA = %.16e\t max(dAdA) = %.16e\n",i, FUNC, DADA, maxdada);
      #ifdef USETEXTURE
       unbind_texture_trafo();
      #endif
      
      //double plaq = calc_plaquette(dev_gf,0);
      //printf("PLAQ = %.16e\n",plaq);
    }
    
    
    
    
    if(maxdada < eps){
      printf("CONVERGENCE!\nFinal values: F = %.16e dAdA = %.16e, maxdAdA = %.16e\n", FUNC, DADA, maxDADA);
     assert((stop = clock())!=-1);
     timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
     printf("Overrelaxation finished after %f sec\n", timeelapsed);      
     #ifdef USETEXTURE
       unbind_texture_gf();
     #endif 
     return(i);
    }

  }//i
 
  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif 

  assert((stop = clock())!=-1);
  timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
  printf("Overrelaxation finished after %f sec\n", timeelapsed);

  
  if(maxdada > eps){
      printf("FAIL! Gauge condition not reached!\nFinal values: F = %.16e dAdA = %.16e, maxdAdA = %.16e\n", FUNC, DADA, maxDADA);
      return(-1);
    }
   else{
     return(i);
   }
}

  
  
  
  
  
void benchmark(){
  
  double timeelapsed = 0.0;
  clock_t start, stop;
  int i;
  int gridsize;


  if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( (VOLUME/2) >= BLOCK){
   gridsize =VOLUME/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 




  assert((start = clock())!=-1);
  #ifdef USETEXTURE
     bind_texture_gf(dev_gf);
  #endif
  
  printf("Doing small benchmark...");
  
for(i=0; i<100; i++){
   // overrelax update
   #ifdef USETEXTURE
    // update of EVEN  
     bind_texture_trafo(dev_trafo1);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo2, dev_gf, dev_trafo1, dev_eoidx_even, dev_eoidx_odd, dev_nn); 
     unbind_texture_trafo();

   // update of ODD
     bind_texture_trafo(dev_trafo2);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo2, dev_eoidx_odd, dev_eoidx_even, dev_nn);
     unbind_texture_trafo();
   
    #else  //USETEXTURE

    // update of EVEN  
     hipFuncSetCacheConfig(reinterpret_cast<const void*>(dev_overrelax_step), hipFuncCachePreferShared);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_even, dev_eoidx_odd, dev_nn); 

   // update of ODD
     hipFuncSetCacheConfig(reinterpret_cast<const void*>(dev_overrelax_step), hipFuncCachePreferShared);
     dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_odd, dev_eoidx_even, dev_nn);  
    #endif  //USETEXTURE
    
}

  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif 

  printf("Done\n"); 
  
  assert((stop = clock())!=-1);
  timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
  
 // x2 because 2x overrelaxstep per iteration
  double benchres = 4766.0*2*(VOLUME/2)* 100 / timeelapsed / 1.0e9;
  printf("Benchmark: %f Gflops\n", benchres); 
   
  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif
}

  
  
  
void calc_star(){
   
  int gridsize;
  double F = 0.0;

  
  if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( (VOLUME/2) >= BLOCK){
   gridsize =VOLUME/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 
  //printf("Setting up GPU configuration: threads/block = %d\t blocks = %d\n", BLOCK, gridsize);
   
   
    
    
   #ifdef USETEXTURE
     bind_texture_gf(dev_gf);
   #endif
   
 /*
   size_t dev_gfsize = 6*VOLUME*sizeof(dev_su3_2v);
   hipMemcpy(h2d_trafo, dev_trafo1, dev_gfsize, hipMemcpyDeviceToHost);

    
   for(i=0; i<10; i++){
    show_su3_2v(&(h2d_trafo[6*i]));
   }
   printf("\n\n"); 
  */
 
  
   //calculate initial F
   F = calc_functional(dev_gf,dev_trafo1);
   
   //calculate star
   
   #ifdef USETEXTURE
     bind_texture_trafo(dev_trafo1);
   #endif
      // update of EVEN 
      dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_even, dev_eoidx_odd, dev_nn); 
   #ifdef USETEXTURE
     unbind_texture_trafo();
   #endif
   

   
  //recalculate F
   F = calc_functional(dev_gf, dev_trafo1);
   
   #ifdef USETEXTURE
     bind_texture_trafo(dev_trafo2);
   #endif
      // update of ODD
      dev_overrelax_step<<< griddim, blockdim >>> (dev_trafo1, dev_gf, dev_trafo1, dev_eoidx_odd, dev_eoidx_even, dev_nn);
   #ifdef USETEXTURE
     unbind_texture_trafo();
   #endif
   
//printf("%s\n", hipGetErrorString(hipGetLastError())); 


   
  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif 
  
}



