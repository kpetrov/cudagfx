#include "hip/hip_runtime.h"


#define MICRO_OVERRELAX_RESOLUTION 1.0e-9

/*


// calculates the staples and stores them in a staples field
//                pos + nu
//  x-------<--------x-------->-------x
//  |                |                |
//  |                |                |
//  |                |                |   
//  v                ^                v
//  |                |                |
//  |                |                |
//  |                |                | 
//  x------->--------x--------<-------x
//                  pos             pos + mu
//
//                  
//                   nu
//                    ^
//                    |
//                     ---> mu 
//
__global__ void dev_get_staples(dev_su3_2v * gf, dev_su3_2v * staples,  int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn){


    int eofieldpos, pos, hoppos, mu, nu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
    
     __shared__ dev_su3 help1[BLOCK];
     __shared__ dev_su3 help2[BLOCK];
     dev_su3 help3, staple;
   
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){

    pos = dev_indeo_thissite[eofieldpos];

    for(mu=0; mu<4; mu++){
      for(nu=0; nu<4; nu++){
        if(mu!=nu){
       
      //staple in forward nu 
          //link in nu dir at (pos + mu)
          hoppos = dev_nn[8*pos + mu];
          #ifdef GF_8
           dev_reconstructgf_8texref(gf, (4*hoppos+nu),&(help1[ix]));
          #else
           dev_reconstructgf_2vtexref(gf, (4*hoppos+nu),&(help1[ix]));
          #endif
          
          //link^+ in mu dir at (pos + nu)
          hoppos = dev_nn[8*pos + nu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*hoppos+mu),&(help2[ix]));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*hoppos+mu),&(help2[ix]));
          #endif
          
          dev_su3_ti_su3(&(help3), &(help1[ix]), &(help2[ix]));

          //link^+ in nu dir at (pos)
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*pos+nu),&(help1[ix]));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*pos+nu),&(help1[ix]));
          #endif
          
          dev_su3_ti_su3(&(staple), &(help3), &(help1[ix]));
      
     //staple in backward nu dir
          //link^+ in nu dir at (pos + mu - nu)
          hoppos = dev_nn[8*pos + mu];
          hoppos = dev_nn[8*hoppos + 4 + nu];
          #ifdef GF_8
           dev_reconstructgf_8texref_dagger(gf, (4*hoppos+nu),&(help1[ix]));
          #else
           dev_reconstructgf_2vtexref_dagger(gf, (4*hoppos+nu),&(help1[ix]));
          #endif
          
          //link^+ in mu dir at (pos - nu)
          hoppos = dev_nn[8*pos + 4 + nu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*hoppos+mu),&(help2[ix]));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*hoppos+mu),&(help2[ix]));
          #endif
          
          dev_su3_ti_su3(&(help3), &(help1[ix]), &(help2[ix]));

          //link in nu dir at (pos - nu)
          hoppos = dev_nn[8*pos + 4 + nu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref(gf, (4*pos+nu),&(help1[ix]));
          #else
            dev_reconstructtrafo_2vtexref(gf, (4*pos+nu),&(help1[ix]));
          #endif
          
          dev_add_su3_ti_su3(&(staple), &(help3), &(help1[ix]));
       
        }//if(mu!=nu)
      }//nu
      
      //now store the staple of the link in mu dir at pos in the staples field 
      //which has size VOLUME/2!!
     #ifdef GF_8
       dev_storegf_8((4*eofieldpos+mu), staples , &(staple) );
     #else
       dev_storegf_2v((4*eofieldpos+mu), staples , &(staple) );
     #endif  
 
    }//mu
  }
}






// do a thermalization gauge heatbath update of either the even or the odd sites depending on the dev_indeo_thissite and dev_indeo_nextside
// index fields

__global__ void dev_gauge_heatbath_sweep (dev_su3_2v * gf_new, dev_su3_2v * staples, dev_su3_2v * gf, int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn, float* rndgauss_field, float* rndunif_field){

    int eofieldpos, pos,mu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 staple[BLOCK];
     dev_su3 help;
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){
    pos = dev_indeo_thissite[eofieldpos];
    
    for(mu=0; mu<4; mu++){
      
      //load u_mu(x)
        #ifdef GF_8
          dev_reconstructgf_8texref(gf, (4*pos+mu),&(gfsmem[ix]));
        #else
          dev_reconstructgf_2vtexref(gf, (4*pos+mu),&(gfsmem[ix]));
        #endif

      //load staple of u_mu(x)
      #ifdef USETEXTURE //workaround for not having to implement yet another texture
        #define USETEXTURE_YES 
      #endif
      #undef USETEXTURE
        #ifdef GF_8
          dev_reconstructgf_8texref(staples, (4*eofieldpos+mu),&(staple[ix]));
        #else
          dev_reconstructgf_2vtexref(staples, (4*eofieldpos+mu),&(staple[ix]));
        #endif
      #ifdef USETEXTURE_YES
        #undef USETEXTURE_YES 
        #define USETEXTURE
      #endif

       dev_su3copy( &(help) , &(gfsmem[ix]) );  
       cabibbo_marinari_heatbath( &(help),  &(staple[ix]) , therm_beta, rndgauss_field, rndunif_field, eofieldpos); 
                    //also append the position of g, for the rng!
   		    //the global sa_beta is used for beta
   								 
     dev_su3_normalize(&(help)); 
   
     #ifdef GF_8
       dev_storegf_8(pos, gf_new ,&(help));
     #else
       dev_storegf_2v(pos, gf_new ,&(help));
     #endif 
   
   }
   
   #ifdef USETEXTURE
    for(mu=0; mu<4; mu++){
     // copy the trafofields of the sites that are not updated to destination field
     // e.g. if EVEN is updated just copy ODD trafos
     pos = dev_indeo_nextside[eofieldpos];
     //load g(x)
     #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+mu) ,&(gfsmem[ix]));
     #else
        dev_reconstructgf_2vtexref(gf, (4*pos+mu) ,&(gfsmem[ix]));
     #endif
     // and store it
     #ifdef GF_8
       dev_storegf_8((4*pos+mu), gf_new ,&(gfsmem[ix]) );
     #else
       dev_storegf_2v((4*pos+mu), gf_new ,&(gfsmem[ix]) );
     #endif    
   }
   #endif
   
   
  }
}

*/








// calculates the staples and stores them in a staples field
//                pos + nu
//  x-------<--------x-------->-------x
//  |                |                |
//  |                |                |
//  |                |                |   
//  v                ^                v
//  |                |                |
//  |                |                |
//  |                |                | 
//  x------->--------x--------<-------x
//                  pos             pos + mu
//
//                  
//                   nu
//                    ^
//                    |
//                     ---> mu 
//
__device__ void dev_get_staple(dev_su3_2v * gf, dev_su3 * staple, int * dev_nn, int pos, int nu){


    int  hoppos, mu;
    // trafo and gauge fields read and reconstructed --> shared mem
    
     dev_su3 help1, help2, help3;
     
     dev_su3zero( staple );

      for(mu=0; mu<4; mu++){
        if(mu!=nu){
       
      //staple in forward nu 
          //link in mu dir at (pos + nu)
          hoppos = dev_nn[8*pos + nu];
          #ifdef GF_8
           dev_reconstructgf_8texref(gf, (4*hoppos+mu),&(help1));
          #else
           dev_reconstructgf_2vtexref(gf, (4*hoppos+mu),&(help1));
          #endif
          
          //link^+ in nu dir at (pos + mu)
          hoppos = dev_nn[8*pos + mu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*hoppos+nu),&(help2));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*hoppos+nu),&(help2));
          #endif
          
          dev_su3_ti_su3(&(help3), &(help1), &(help2));

          //link^+ in mu dir at (pos)
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*pos+mu),&(help1));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*pos+mu),&(help1));
          #endif
          
          dev_add_su3_ti_su3(staple, &(help3), &(help1));
      
     //staple in backward mu dir
          //link^+ in mu dir at (pos + nu - mu)
          hoppos = dev_nn[8*pos + nu];
          hoppos = dev_nn[8*hoppos + 4 + mu];
          #ifdef GF_8
           dev_reconstructgf_8texref_dagger(gf, (4*hoppos+mu),&(help1));
          #else
           dev_reconstructgf_2vtexref_dagger(gf, (4*hoppos+mu),&(help1));
          #endif
          
          //link^+ in nu dir at (pos - mu)
          hoppos = dev_nn[8*pos + 4 + mu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref_dagger(gf, (4*hoppos+nu),&(help2));
          #else
            dev_reconstructtrafo_2vtexref_dagger(gf, (4*hoppos+nu),&(help2));
          #endif
          
          dev_su3_ti_su3(&(help3), &(help1), &(help2));

          //link in mu dir at (pos - mu)
          hoppos = dev_nn[8*pos + 4 + mu];
          #ifdef GF_8
            dev_reconstructtrafo_8texref(gf, (4*hoppos+mu),&(help1));
          #else
            dev_reconstructtrafo_2vtexref(gf, (4*hoppos+mu),&(help1));
          #endif
          
          dev_add_su3_ti_su3(staple, &(help3), &(help1));
       
        }//if(mu!=nu)
      }//nu 
}









__device__ void dev_microorx_su2(dev_su2 * out, dev_su2 * in){

  double Tr_WW = (*in).a.x*(*in).a.x + (*in).a.y*(*in).a.y + (*in).b.x*(*in).b.x +  (*in).b.y*(*in).b.y;
  double fac;
  
  if(abs(Tr_WW) < MICRO_OVERRELAX_RESOLUTION){
     (*out).a.x = 1.0;
     (*out).a.y = 0.0;
     (*out).b.x = 0.0;
     (*out).b.y = 0.0;
  }
  else{
    fac = 2.0*(*in).a.x/ Tr_WW;
    (*out).a.x = fac*(*in).a.x;
    (*out).a.y = fac*(*in).a.y;
    (*out).b.x = fac*(*in).b.x;
    (*out).b.y = fac*(*in).b.y;
    (*out).a.x = (*out).a.x - 1.0;
  }

}


__device__ void cabibbo_marinari_microorx(dev_su3 * g, dev_su3 * staple){
// this routine does an microoverrelaxation update of an SU(3) matrix g


  int a,b,c;
  dev_su3 X;
  dev_su2 w, alpha;
  dev_complex dummy, dummy2, dummy3;

  
    for(a=0; a<2; a++){
      for(b=a+1; b<3; b++){
      
      dev_su3_ti_su3(&(X), g, staple);
      
  
  w.a.x = X[a][a].re + X[b][b].re;
  w.b.y = -X[a][a].im + X[b][b].im;
  w.a.y = -X[a][b].im - X[b][a].im; 
  w.b.x = -X[a][b].re + X[b][a].re;
  
 
  
  /*
     my definitions: U = w0 ID + i ( w1 sigma1 + w2 sigma2 + w3 sigma3 )
     
               | 0  1 |           | 0  -i |            | 1  0 |
     sigma1 =  |      |  sigma2 = |       |   sigma3 = |      |
               | 1  0 |           | i   0 |            | 0 -1 |
  */
  
    
      dev_microorx_su2(&(alpha), &(w));  
       
        
      for(c=0; c<3; c++){
      
        
      //dummy  =  cmplx(alpha(0),alpha(3), kind=RKIND) * u(a,c)     &
      //             + cmplx(alpha(2),alpha(1), kind=RKIND) * u(b,c)

       dummy = dev_cmult(dev_initcomplex(alpha.a.x,alpha.b.y),(*g)[a][c]);
       dummy2 = dev_cmult(dev_initcomplex(alpha.b.x,alpha.a.y),(*g)[b][c]);
       dummy = dev_cadd(dummy, dummy2);
       
       //     u(b,c) =  cmplx(-alpha(2), alpha(1), kind=RKIND) * u(a,c)   &
       //             + cmplx( alpha(0),-alpha(3), kind=RKIND) * u(b,c)
       
       dummy2 = dev_cmult(dev_initcomplex(-alpha.b.x,alpha.a.y),(*g)[a][c]);
       dummy3 = dev_cmult(dev_initcomplex(alpha.a.x,-alpha.b.y),(*g)[b][c]); 
       (*g)[b][c] = dev_cadd(dummy2, dummy3);
       
       //     u(a,c) = dummy   
       
       (*g)[a][c] = dummy;
       
      } 
         
    }
  } 
}






// do a thermalization gauge microoverrelax update of either the even or the odd sites depending on the dev_indeo_thissite and dev_indeo_nextside
// index fields

__global__ void dev_gauge_microorx_sweep (dev_su3_2v * gf_new, dev_su3_2v * staples, dev_su3_2v * gf, int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn, float* rndgauss_field, float* rndunif_field, int nu){

    int eofieldpos, pos;
    // trafo and gauge fields read and reconstructed --> shared mem
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 staple[BLOCK];
     dev_su3 help;
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){
    pos = dev_indeo_thissite[eofieldpos];
    
    //for(mu=0; mu<4; mu++){
      
      //load u_mu(x)
        #ifdef GF_8
          dev_reconstructgf_8texref(gf, (4*pos+nu),&(gfsmem[ix]));
        #else
          dev_reconstructgf_2vtexref(gf, (4*pos+nu),&(gfsmem[ix]));
        #endif

      //construct staple of u_mu(x)
      dev_get_staple(gf, &(staple[ix]), dev_nn, pos, nu);
      
      dev_su3copy( &(help) , &(gfsmem[ix]) );  
      cabibbo_marinari_microorx( &(help),  &(staple[ix]));
      
       //also append the position of g, for the rng!
       //the global sa_beta is used for beta
   								 
     dev_su3_normalize(&(help)); 
   
     #ifdef GF_8
       dev_storegf_8(pos, gf_new ,&(help));
     #else
       dev_storegf_2v(pos, gf_new ,&(help));
     #endif 
   
   //}
   
   #ifdef USETEXTURE
    //for(mu=0; mu<4; mu++){
     // copy the trafofields of the sites that are not updated to destination field
     // e.g. if EVEN is updated just copy ODD trafos
     pos = dev_indeo_nextside[eofieldpos];
     //load g(x)
     #ifdef GF_8
        dev_reconstructgf_8texref(gf, (4*pos+nu) ,&(gfsmem[ix]));
     #else
        dev_reconstructgf_2vtexref(gf, (4*pos+nu) ,&(gfsmem[ix]));
     #endif
     // and store it
     #ifdef GF_8
       dev_storegf_8((4*pos+nu), gf_new ,&(gfsmem[ix]) );
     #else
       dev_storegf_2v((4*pos+nu), gf_new ,&(gfsmem[ix]) );
     #endif    
   //}
   #endif
   
  }
}















// do a thermalization gauge heatbath update of either the even or the odd sites depending on the dev_indeo_thissite and dev_indeo_nextside
// index fields

__global__ void dev_gauge_heatbath_sweep (dev_su3_2v * gf_new, dev_su3_2v * staples, dev_su3_2v * gf, int * dev_indeo_thissite, int * dev_indeo_nextside, int * dev_nn, float* rndgauss_field, float* rndunif_field, int nu){

    int eofieldpos, pos;
    // trafo and gauge fields read and reconstructed --> shared mem
    
     __shared__ dev_su3 gfsmem[BLOCK];
     __shared__ dev_su3 staple[BLOCK];
     dev_su3 help;
    
  eofieldpos = threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(eofieldpos < dev_VOLUME/2){
    pos = dev_indeo_thissite[eofieldpos];
     
      //load u_nu(x)
        #ifdef GF_8
          dev_reconstructgf_8texref(gf, (4*pos+nu),&(gfsmem[ix]));
        #else
          dev_reconstructgf_2vtexref(gf, (4*pos+nu),&(gfsmem[ix]));
        #endif

      //construct staple of u_nu(x)
      dev_get_staple(gf, &(staple[ix]), dev_nn, pos, nu);
      
      dev_su3copy( &(help) , &(gfsmem[ix]) );  
      cabibbo_marinari_heatbath( &(help),  &(staple[ix]) , therm_beta, rndgauss_field, rndunif_field, 4*eofieldpos+nu); 
                    //also append the position of g, for the rng!
   		    //the global sa_beta is used for beta
   								 
     dev_su3_normalize(&(help)); 
   
     #ifdef GF_8
       dev_storegf_8(pos, gf_new ,&(help));
     #else
       dev_storegf_2v(pos, gf_new ,&(help));
     #endif 
   
   
   #ifdef USETEXTURE
     // copy the trafofields of the sites that are not updated to destination field
     // e.g. if EVEN is updated just copy ODD trafos
     pos = dev_indeo_nextside[eofieldpos];
     //load g(x)
     #ifdef GF_8
        dev_reconstructgf_8texref(gf, pos ,&(gfsmem[ix]));
     #else
        dev_reconstructgf_2vtexref(gf, pos ,&(gfsmem[ix]));
     #endif
     // and store it
     #ifdef GF_8
       dev_storegf_8(pos, gf_new ,&(gfsmem[ix]) );
     #else
       dev_storegf_2v(pos, gf_new ,&(gfsmem[ix]) );
     #endif    
   #endif
   
  }
}






void set_beta(double b){
   double beta;
   beta = b/3.0; // -> beta = b/3 for NCOL!!  
   //set this beta on device
   CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL("therm_beta"), &beta, sizeof(double)) ) ;
}




// perform thermalization (simulation) of gauge field
void thermalize_gauge(){

  int gridsize, mu, imicro;
  double plaq;
  int i;
  clock_t start, stop; 
  double timeelapsed = 0.0;
  hipError_t cudaerr;
  
  if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( (VOLUME/2) >= BLOCK){
   gridsize =VOLUME/2/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1); 
  
  printf("Kernel parameter: gridsize = %d, blocksize = %d, V = %d\n", gridsize, BLOCK, gridsize*BLOCK);
  
  
  FILE * plaqfile = fopen("plaq.txt", "w");
  if(!plaqfile){
    fprintf(stderr, "Could not open file 'plaq.txt'\n");
  }
  
// Start timer
  assert((start = clock())!=-1);  
   
 //set the beta
 printf("Setting beta to %f\n", thermparam.beta);
 set_beta(thermparam.beta);
 
 for(i=0; i<thermparam.Nsweep; i++){
   
   //printf("Doing microorx sweep...\n");
  hipDeviceSynchronize();
   cudaerr = hipGetLastError();
   if(cudaerr != hipSuccess){
     printf("%s\n", hipGetErrorString(cudaerr)); 
   } 
   // microorx update
   

for(imicro=0; imicro<0; imicro++){  // ***************** MICRO_OVERRELAXATION
#ifdef USETEXTURE
  for(mu=0; mu<4; mu++){
   // update of EVEN  
      bind_texture_gf(dev_gf);
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */  
      dev_gauge_microorx_sweep<<< griddim, blockdim >>> (dev_gf2, dev_staples,
       			  dev_gf, dev_eoidx_even, dev_eoidx_odd, 
                          dev_nn, dev_rndgauss_field,dev_rndunif_field, mu); 
                                                
                                                 
      unbind_texture_gf();
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }
  // update of ODD
      bind_texture_gf(dev_gf2);
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf2, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */

      dev_gauge_microorx_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples, dev_gf2, 
                                   dev_eoidx_odd, dev_eoidx_even, dev_nn,
                                   dev_rndgauss_field, dev_rndunif_field, mu);    
                                                                                                    
      unbind_texture_gf();
     
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }   
  }//mu       
#else  //USETEXTURE
  for(mu=0; mu<4; mu++){
   // update of EVEN  
       /*
       dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
       */

       dev_gauge_microorx_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples,
                                   dev_gf,  dev_eoidx_even, dev_eoidx_odd, 
                                   dev_nn, dev_rndgauss_field, dev_rndunif_field, mu);

      hipDeviceSynchronize();  
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }  
    
  // update of ODD
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */
      dev_gauge_microorx_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples, dev_gf, 
                                 dev_eoidx_odd, dev_eoidx_even, dev_nn,
                                 dev_rndgauss_field, dev_rndunif_field, mu);
                                                
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      } 
   }//mu
#endif  //USETEXTURE
    
}//imicro                          // ***************** MICRO_OVERRELAXATION



   
   hipDeviceSynchronize();
   cudaerr = hipGetLastError();
   if(cudaerr != hipSuccess){
     printf("%s\n", hipGetErrorString(cudaerr)); 
   } 
                                   // ********************  HEATBATH
 
#ifdef USETEXTURE
  for(mu=0; mu<4; mu++){
   // update of EVEN  
      bind_texture_gf(dev_gf);
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */  
      dev_gauge_heatbath_sweep<<< griddim, blockdim >>> (dev_gf2, dev_staples,
       			  dev_gf, dev_eoidx_even, dev_eoidx_odd, 
                          dev_nn, dev_rndgauss_field,dev_rndunif_field, mu); 
                                                
                                                 
      unbind_texture_gf();
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }
     
     
      // generate new random numbers
      //printf("Updating the random numbers...\n");
      hipGetLastError();
      update_MT();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }     
     
  // update of ODD
      bind_texture_gf(dev_gf2);
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf2, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */

      dev_gauge_heatbath_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples, dev_gf2, 
                                   dev_eoidx_odd, dev_eoidx_even, dev_nn,
                                   dev_rndgauss_field, dev_rndunif_field, mu);    
                                                                                                    
      unbind_texture_gf();
     
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }   

      // generate new random numbers
      //printf("Updating the random numbers...\n");
      hipGetLastError();
      update_MT();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }   
  }//mu       
#else  //USETEXTURE
  for(mu=0; mu<4; mu++){
   // update of EVEN  
       /*
       dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
       */

       dev_gauge_heatbath_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples,
                                   dev_gf,  dev_eoidx_even, dev_eoidx_odd, 
                                   dev_nn, dev_rndgauss_field, dev_rndunif_field, mu);

      hipDeviceSynchronize();  
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }


      // generate new random numbers
      //printf("Updating the random numbers...\n");
      hipGetLastError();
      update_MT();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }    
    
  // update of ODD
      /*
      dev_get_staples<<< griddim, blockdim >>>(dev_gf, dev_staples, 
                                       dev_eoidx_even, dev_eoidx_odd, dev_nn);
      */
      dev_gauge_heatbath_sweep<<< griddim, blockdim >>> (dev_gf, dev_staples, dev_gf, 
                                 dev_eoidx_odd, dev_eoidx_even, dev_nn,
                                 dev_rndgauss_field, dev_rndunif_field, mu);
                                                
      hipDeviceSynchronize();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }

      // generate new random numbers
      //printf("Updating the random numbers...\n");
      hipGetLastError();
      update_MT();
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }     
   }//mu
#endif  //USETEXTURE                     // ********************  HEATBATH
    

    if((i%thermparam.checkint) == 0){    

      hipGetLastError();
      plaq = calc_plaquette(dev_gf,0);
      cudaerr = hipGetLastError();
      if(cudaerr != hipSuccess){
        printf("%s\n", hipGetErrorString(cudaerr)); 
      }
      
      //plaq = 6.0*(1.0-plaq);
      fprintf(plaqfile, "%d %.16e\n", i, plaq);
      printf("therm %6d:\t PLAQ = %.16e\n",i, plaq);
    }
   

  }//i
 
  #ifdef USETEXTURE
    unbind_texture_gf();
  #endif 
  
  
  assert((stop = clock())!=-1);
  timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
  fclose(plaqfile);
  printf("Thermalization finished after %f sec\n", timeelapsed); 
}








