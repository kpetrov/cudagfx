#include "hip/hip_runtime.h"



 /* texture for gauge field */
 texture<int4,1> gf_tex;
 const textureReference* gf_texRefPtr = NULL;
 hipChannelFormatDesc gf_channelDesc;
 
 /* texture for trafo field */
 texture<int4,1> trafo_tex;
 const textureReference* trafo_texRefPtr = NULL;
 hipChannelFormatDesc trafo_channelDesc;




__device__ inline dev_complex dev_cconj (dev_complex c){ /*konjugiert komplexe Zahl*/
 dev_complex erg;
 erg.re = c.re;
 erg.im = -1.0*c.im;
return erg;
}

__device__ inline void dev_ccopy(dev_complex* von, dev_complex* nach){/*kopiert complex von nach complex nach*/
  nach->re = von->re;
  nach->im = von->im;
}

__device__ inline double dev_cabssquare (dev_complex c){ /*gibt abs^2 einer komplexen Zahl zurück*/
 return c.re*c.re + c.im*c.im;
}

__device__ inline double dev_cabsolute (dev_complex c){/*gibt Betrag einer kompl. zahl zurück*/
 return sqrt(c.re*c.re + c.im*c.im);
}



__device__ inline  dev_complex dev_crealmult(dev_complex c1, double real){ /*multipliziert c1 mit reeller zahl re*/
  dev_complex erg;
  erg.re = real*c1.re;
  erg.im = real*c1.im;
return erg;
}

__device__ inline dev_complex dev_cmult (dev_complex c1, dev_complex c2){ /*multiplizier zwei komplexe Zahlen*/
  dev_complex erg;
  erg.re = c1.re * c2.re - c1.im * c2.im;
  erg.im = c1.re * c2.im + c1.im * c2.re;
return erg;
}

__device__ inline dev_complex dev_cadd (dev_complex c1, dev_complex c2){ /*addiert zwei komplexe Zahlen */
  dev_complex erg;
  erg.re = c1.re + c2.re;
  erg.im = c1.im + c2.im;
return erg;
}


__device__ inline dev_complex dev_cdiv(dev_complex c1, dev_complex c2) { /* dividiert c1 durch c2 */
  dev_complex erg;
  double oneovernenner = 1.0/(c2.re*c2.re + c2.im*c2.im);
  erg.re = oneovernenner*(c1.re*c2.re + c1.im*c2.im);
  erg.im = oneovernenner*(c1.im*c2.re - c1.re*c2.im);
return erg;
}


__device__ inline dev_complex dev_csub(dev_complex c1, dev_complex c2){
   dev_complex erg;
   erg.re = c1.re - c2.re;
   erg.im = c1.im - c2.im;
return erg;
}


__device__ inline dev_complex dev_initcomplex(double re, double im){/* gibt komplexe Zahl mit Realt re und Imt im zurück*/
    dev_complex erg;
    erg.re = re;
    erg.im = im;
return (erg);
}



__device__ void dev_unitsu3(dev_su3 * g){
  (*g)[0][0].re = 1.0;
  (*g)[0][0].im = 0.0;
  (*g)[0][1].re = 0.0;
  (*g)[0][1].im = 0.0;
  (*g)[0][2].re = 0.0;
  (*g)[0][2].im = 0.0;
  
  (*g)[1][0].re = 0.0;
  (*g)[1][0].im = 0.0;
  (*g)[1][1].re = 1.0;
  (*g)[1][1].im = 0.0;
  (*g)[1][2].re = 0.0;
  (*g)[1][2].im = 0.0; 
   
  (*g)[2][0].re = 0.0;
  (*g)[2][0].im = 0.0;
  (*g)[2][1].re = 0.0;
  (*g)[2][1].im = 0.0;
  (*g)[2][2].re = 1.0;
  (*g)[2][2].im = 0.0;
   
}








__device__ void dev_storetrafo_2v(int pos, dev_su3_2v* trafofield , dev_su3* g){

   trafofield[6*pos].x = (*g)[0][0].re;
   trafofield[6*pos].y = (*g)[0][0].im;
   trafofield[6*pos+1].x = (*g)[0][1].re;
   trafofield[6*pos+1].y = (*g)[0][1].im;
   
   trafofield[6*pos+2].x = (*g)[0][2].re;
   trafofield[6*pos+2].y = (*g)[0][2].im;
   trafofield[6*pos+3].x = (*g)[1][0].re;
   trafofield[6*pos+3].y = (*g)[1][0].im;
   
   trafofield[6*pos+4].x = (*g)[1][1].re;
   trafofield[6*pos+4].y = (*g)[1][1].im;
   trafofield[6*pos+5].x = (*g)[1][2].re;
   trafofield[6*pos+5].y = (*g)[1][2].im;
   
}


__device__ void dev_storegf_2v(int pos, dev_su3_2v* gfield , dev_su3* U){

   gfield[6*pos].x = (*U)[0][0].re;
   gfield[6*pos].y = (*U)[0][0].im;
   gfield[6*pos+1].x = (*U)[0][1].re;
   gfield[6*pos+1].y = (*U)[0][1].im;
   
   gfield[6*pos+2].x = (*U)[0][2].re;
   gfield[6*pos+2].y = (*U)[0][2].im;
   gfield[6*pos+3].x = (*U)[1][0].re;
   gfield[6*pos+3].y = (*U)[1][0].im;
   
   gfield[6*pos+4].x = (*U)[1][1].re;
   gfield[6*pos+4].y = (*U)[1][1].im;
   gfield[6*pos+5].x = (*U)[1][2].re;
   gfield[6*pos+5].y = (*U)[1][2].im;
   
}




__device__ void dev_storegf_8(int pos, dev_su3_2v* trafofield , dev_su3* U){


   // a2, a3
    trafofield[4*pos].x   = (*U)[0][1].re;
    trafofield[4*pos].y   = (*U)[0][1].im;
    trafofield[4*pos+1].x = (*U)[0][2].re;
    trafofield[4*pos+1].y = (*U)[0][2].im;
    
   // theta_a1, theta_c1
   // use atan2 for this: following the reference, atan2 should give an angle -pi < phi < +pi  
   trafofield[4*pos+2].x = ( atan2((*U)[0][0].im, (*U)[0][0].re ));
   trafofield[4*pos+2].y = ( atan2((*U)[2][0].im, (*U)[2][0].re ));
     
   // b1
    trafofield[4*pos+3].x = (*U)[1][0].re ;
    trafofield[4*pos+3].y = (*U)[1][0].im ;

}



__device__ void dev_storetrafo_8(int pos, dev_su3_2v* gfield , dev_su3* g){


   // a2, a3
    gfield[4*pos].x   = (*g)[0][1].re;
    gfield[4*pos].y   = (*g)[0][1].im;
    gfield[4*pos+1].x = (*g)[0][2].re;
    gfield[4*pos+1].y = (*g)[0][2].im;
    
   // theta_a1, theta_c1
   // use atan2 for this: following the reference, atan2 should give an angle -pi < phi < +pi  
   gfield[4*pos+2].x = ( atan2((*g)[0][0].im, (*g)[0][0].re ));
   gfield[4*pos+2].y = ( atan2((*g)[2][0].im, (*g)[2][0].re ));
     
   // b1
    gfield[4*pos+3].x = (*g)[1][0].re ;
    gfield[4*pos+3].y = (*g)[1][0].im ;

}







__inline__ __device__ double2 tex1Dfetch_gf(const int& i)
{
int4 v=tex1Dfetch(gf_tex, i);
return make_double2(__hiloint2double(v.y, v.x),__hiloint2double(v.w, v.z));
}

__inline__ __device__ double2 tex1Dfetch_trafo(const int& i)
{
int4 v=tex1Dfetch(trafo_tex, i);
return make_double2(__hiloint2double(v.y, v.x),__hiloint2double(v.w, v.z));
}




// reconstruction of the link fields from two rows of the su3 matrix
// numbers are fetched from texture cache
__device__ void dev_reconstructgf_2vtexref (dev_su3_2v * field, int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  double2 gfin;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos);
    gfin = tex1Dfetch_gf(6*pos);
  #else
    gfin = field[6*pos];
  #endif  
  
  //first row
  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = gfin.y;
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+1);
    gfin = tex1Dfetch_gf(6*pos+1);
  #else
    gfin = field[6*pos+1];
  #endif   
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;
  
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+2);
    gfin = tex1Dfetch_gf(6*pos+2);
  #else
    gfin = field[6*pos+2];
  #endif  
  (*gf)[0][2].re = gfin.x;
  (*gf)[0][2].im = gfin.y;
  
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+3);
    gfin = tex1Dfetch_gf(6*pos+3);
  #else
    gfin = field[6*pos+3];
  #endif    
  //second row
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = gfin.y;
    
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+4);
    gfin = tex1Dfetch_gf(6*pos+4);
  #else
    gfin = field[6*pos+4];
  #endif  
  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+5);
    gfin = tex1Dfetch_gf(6*pos+5);
  #else
    gfin = field[6*pos+5];
  #endif    
  (*gf)[1][2].re = gfin.x;
  (*gf)[1][2].im = gfin.y;
  
  //third row from cconj(cross product of first and second row)
  help1 = dev_cmult((*gf)[0][1],(*gf)[1][2]);
  help2 = dev_cmult((*gf)[0][2],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][0] = help1;

  
  help1 = dev_cmult((*gf)[0][2],(*gf)[1][0]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[1][2]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][1] = help1;
  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[0][1],(*gf)[1][0]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][2] = help1;
  
  return;
}




// su3 - dagger reconstruction from two rows  
__device__ void dev_reconstructgf_2vtexref_dagger (dev_su3_2v * field, int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  double2 gfin;
  
  
  //first column (minus in im for complex conj.)

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos);
    gfin = tex1Dfetch_gf(6*pos);
  #else
    gfin = field[6*pos];
  #endif  
  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+1);
    gfin = tex1Dfetch_gf(6*pos+1);
  #else
    gfin = field[6*pos+1];
  #endif    
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = -gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+2);
    gfin = tex1Dfetch_gf(6*pos+2);
  #else
    gfin = field[6*pos+2];
  #endif  
  (*gf)[2][0].re = gfin.x;
  (*gf)[2][0].im = -gfin.y;


  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+3);
    gfin = tex1Dfetch_gf(6*pos+3);
  #else
    gfin = field[6*pos+3];
  #endif    
  //second  column (minus in im for complex conj.)
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+4);
    gfin = tex1Dfetch_gf(6*pos+4);
  #else
    gfin = field[6*pos+4];
  #endif  
  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,6*pos+5);
    gfin = tex1Dfetch_gf(6*pos+5);
  #else
    gfin = field[6*pos+5];
  #endif    
  (*gf)[2][1].re = gfin.x;
  (*gf)[2][1].im = -gfin.y;
  
  //third column from (cross product of cconj(first column) and cconj(second column))
 
  help1 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[2][1]));
  help2 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[1][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[0][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[0][1]));
  help2 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[2][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[1][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[1][1]));
  help2 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[2][2] = help1;
  
  
  /* does this also work?
  help1 = dev_cmult((*gf)[1][0],(*gf)[2][1]);
  help2 = dev_cmult((*gf)[2][0],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[0][2] = help1;

  
  help1 = dev_cmult((*gf)[2][0],(*gf)[0][1]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[2][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[1][2] = help1;

  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][2] = help1;
  */
}








// reconstruction of the gf using 8 real parameters as 
// described in the appendix of hep-lat 0911.3191 (M.Clark et al.)
// optimized once
__device__ void dev_reconstructgf_8texref (dev_su3_8 * field, int pos, dev_su3* gf){

  double2 gfin, gfin2;
  double one_over_N, help;
  dev_complex p1,p2;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,4*pos);
    gfin = tex1Dfetch_gf(4*pos);
  #else
    gfin = field[4*pos];
  #endif  
  // read a2 a3
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;
  
  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(gf_tex,4*pos+1);
    gfin2 = tex1Dfetch_gf(4*pos+1);
  #else
    gfin2 = field[4*pos+1];
  #endif    
  (*gf)[0][2].re = gfin2.x;
  (*gf)[0][2].im = gfin2.y;  
 
  p1.re = gfin.x*gfin.x + gfin.y*gfin.y + gfin2.x*gfin2.x + gfin2.y*gfin2.y; // use later on
  one_over_N = rsqrt(p1.re); //reciprocal sqrt

  // read theta_a1, theta_c1, b1
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,4*pos+2);
    gfin = tex1Dfetch_gf(4*pos+2);
  #else
    gfin = field[4*pos+2];
  #endif  

  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(gf_tex,4*pos+3);
    gfin2 = tex1Dfetch_gf(4*pos+3);
  #else
    gfin2 = field[4*pos+3];
  #endif 
  
  // reconstruct a1 use sqrt instead of sin
  help = 1.0 - p1.re;
  if(help > 0.0){
     p1.re = sqrtf(help);
  }
  else{
    p1.re = 0.0;
  }

  
  sincos(gfin.x, &(*gf)[0][0].im, &(*gf)[0][0].re);
  (*gf)[0][0].re = (*gf)[0][0].re * p1.re;
  (*gf)[0][0].im = (*gf)[0][0].im * p1.re;
  
  
  
  // assign b1
  (*gf)[1][0].re = gfin2.x;
  (*gf)[1][0].im = gfin2.y;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[1][0].re;
  p2.im = one_over_N*(*gf)[1][0].im;  


  // reconstruct c1 use sqrt instead of sin
  help =1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[1][0].re * (*gf)[1][0].re - (*gf)[1][0].im * (*gf)[1][0].im;
  if(help > 0.0){
    p1.re = sqrtf(help);
  }   
  else{      
    p1.re = 0.0;  
  }
  sincos(gfin.y, &(*gf)[2][0].im, &(*gf)[2][0].re);
  (*gf)[2][0].re = (*gf)[2][0].re * p1.re; 
  (*gf)[2][0].im = (*gf)[2][0].im * p1.re;
   
  
  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[2][0].re;
  p1.im = - one_over_N*(*gf)[2][0].im;
  
  
  
  //use the last reconstructed gf component gf[2][2] (c3) as a help variable for b2,b3 and c2
  //this is in order to save registers and to prevent extra loading and storing from global mem
  // calculate b2
  
  (*gf)[1][1].re = p1.re*(*gf)[0][2].re;
  (*gf)[1][1].re += p1.im*(*gf)[0][2].im;
  (*gf)[1][1].im = p1.im*(*gf)[0][2].re;
  (*gf)[1][1].im -= p1.re*(*gf)[0][2].im;
  
  (*gf)[2][2].re = (*gf)[0][0].re * (*gf)[0][1].re;
  (*gf)[2][2].re += (*gf)[0][0].im * (*gf)[0][1].im;
  
  (*gf)[2][2].im = (*gf)[0][0].re * (*gf)[0][1].im;
  (*gf)[2][2].im -= (*gf)[0][0].im * (*gf)[0][1].re;
  (*gf)[2][2] = dev_cmult(p2, (*gf)[2][2]);
  
  (*gf)[1][1].re = -one_over_N*( (*gf)[1][1].re + (*gf)[2][2].re);
  (*gf)[1][1].im = -one_over_N*((*gf)[1][1].im + (*gf)[2][2].im);
  
  
  
  
  
  // calculate b3
  (*gf)[1][2].re = p1.re*(*gf)[0][1].re;
  (*gf)[1][2].re += p1.im*(*gf)[0][1].im;
  (*gf)[1][2].im = p1.im*(*gf)[0][1].re;
  (*gf)[1][2].im -= p1.re*(*gf)[0][1].im;
  
  (*gf)[2][2].re = (*gf)[0][0].re*(*gf)[0][2].re;
  (*gf)[2][2].re += (*gf)[0][0].im*(*gf)[0][2].im;
  (*gf)[2][2].im = (*gf)[0][0].re*(*gf)[0][2].im;
  (*gf)[2][2].im -= (*gf)[0][0].im*(*gf)[0][2].re;
  (*gf)[2][2] = dev_cmult(p2,(*gf)[2][2]);
  
  (*gf)[1][2].re = one_over_N*( (*gf)[1][2].re - (*gf)[2][2].re);
  (*gf)[1][2].im = one_over_N*( (*gf)[1][2].im - (*gf)[2][2].im);
  
  
  // calculate c2
  (*gf)[2][1].re = p2.re*(*gf)[0][2].re;
  (*gf)[2][1].re -= p2.im*(*gf)[0][2].im;
  (*gf)[2][1].im = -p2.re*(*gf)[0][2].im;
  (*gf)[2][1].im -= p2.im*(*gf)[0][2].re;
  
  

  (*gf)[2][2].re = (*gf)[0][0].re*(*gf)[0][1].re;
  (*gf)[2][2].re += (*gf)[0][0].im*(*gf)[0][1].im;
  (*gf)[2][2].im = (*gf)[0][0].re* (*gf)[0][1].im;
  (*gf)[2][2].im -= (*gf)[0][0].im* (*gf)[0][1].re;
  help = (*gf)[2][2].re;
  (*gf)[2][2].re = p1.re*(*gf)[2][2].re;
  (*gf)[2][2].re += p1.im*(*gf)[2][2].im;
  (*gf)[2][2].im = p1.re*(*gf)[2][2].im - p1.im*help;
  
  
  (*gf)[2][1].re = one_over_N*((*gf)[2][1].re - (*gf)[2][2].re);
  (*gf)[2][1].im = one_over_N*((*gf)[2][1].im - (*gf)[2][2].im);
  
  // now we have to use p2 and p1 as a help variable, as this is not 
  // needed any more after the first
  // step
  // calculate c3
  (*gf)[2][2].re = p2.re * (*gf)[0][1].re;
  (*gf)[2][2].re -= p2.im * (*gf)[0][1].im;
  (*gf)[2][2].im = - p2.im*(*gf)[0][1].re;
  (*gf)[2][2].im -= p2.re*(*gf)[0][1].im;
  
  p2.re = (*gf)[0][0].re * (*gf)[0][2].re;
  p2.re += (*gf)[0][0].im * (*gf)[0][2].im;
  p2.im = (*gf)[0][0].re * (*gf)[0][2].im;
  p2.im -= (*gf)[0][0].im * (*gf)[0][2].re;
  p2 = dev_cmult(  dev_cconj(p1) , p2);
  
  (*gf)[2][2] = dev_cadd((*gf)[2][2], p2);
  (*gf)[2][2] = dev_crealmult((*gf)[2][2], -one_over_N);
                      
}






__device__ void dev_reconstructgf_8texref_dagger (dev_su3_8 * field, int pos, dev_su3* gf){


  double2 gfin, gfin2;
  double one_over_N, help;
  dev_complex p1,p2;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,4*pos);
    gfin = tex1Dfetch_gf(4*pos);
  #else
    gfin = field[4*pos];
  #endif 
  // read a2 a3
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(gf_tex,4*pos+1);
    gfin2 = tex1Dfetch_gf(4*pos+1);
  #else
    gfin2 = field[4*pos+1];
  #endif   
  (*gf)[2][0].re = gfin2.x;
  (*gf)[2][0].im = -gfin2.y;  
 
  p1.re = gfin.x*gfin.x + gfin.y*gfin.y + gfin2.x*gfin2.x + gfin2.y*gfin2.y; // use later on
  one_over_N = rsqrt(p1.re);  // reciprocal sqrt

  
  // read theta_a1, theta_c1, b1
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(gf_tex,4*pos+2);
    gfin = tex1Dfetch_gf(4*pos+2);
  #else
    gfin = field[4*pos+2];
  #endif 

  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(gf_tex,4*pos+3);
    gfin2 = tex1Dfetch_gf(4*pos+3);
  #else
    gfin2 = field[4*pos+3];
  #endif 
  
  // reconstruct a1
  help = 1.0 - p1.re;
  if(help > 0.0){
     p1.re = sqrtf(help);   
  }
  else{
    p1.re = 0.0;
  }
  //(*gf)[0][0].re = p1.re*cos(gfin.x);
  //(*gf)[0][0].im = -p1.re*sin(gfin.x);
  
  sincos(gfin.x, &(*gf)[0][0].im, &(*gf)[0][0].re);
  (*gf)[0][0].re = (*gf)[0][0].re * p1.re;
  (*gf)[0][0].im = -(*gf)[0][0].im * p1.re;
    
  
  
  
  // assign b1
  (*gf)[0][1].re = gfin2.x;
  (*gf)[0][1].im = -gfin2.y;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[0][1].re;
  p2.im = -one_over_N*(*gf)[0][1].im;  


  // reconstruct c1
  help = 1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[0][1].re * (*gf)[0][1].re - (*gf)[0][1].im * (*gf)[0][1].im;
  if(help > 0.0){
    p1.re = sqrtf(help);
  }
  else{
    p1.re = 0.0;
  }
  //(*gf)[0][2].re = p1.re*cos(gfin.y);
  //(*gf)[0][2].im = -p1.re*sin(gfin.y);
  
  sincos(gfin.y, &(*gf)[0][2].im, &(*gf)[0][2].re);
  (*gf)[0][2].re = (*gf)[0][2].re * p1.re;
  (*gf)[0][2].im = -(*gf)[0][2].im * p1.re;
     
  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[0][2].re;
  p1.im = one_over_N*(*gf)[0][2].im;
  
  //use the last reconstructed gf component gf[2][2] (c3) as a help variable for b2,b3 and c2
  //this is in order to save registers and to prevent extra loading and storing from global mem
  // calculate b2
  (*gf)[1][1] = dev_cmult(p1,   (*gf)[2][0]    );
  (*gf)[2][2] = dev_cmult(p2, dev_cmult( (*gf)[0][0] , dev_cconj((*gf)[1][0] ))  );
  (*gf)[1][1] = dev_cadd((*gf)[1][1], (*gf)[2][2]);
  (*gf)[1][1] = dev_cconj(dev_crealmult((*gf)[1][1], -one_over_N));
  
  // calculate b3
  (*gf)[2][1] = dev_cmult(p1,   (*gf)[1][0]    );
  (*gf)[2][2] = dev_cmult(p2, dev_cmult( (*gf)[0][0] , dev_cconj((*gf)[2][0] ))  );
  (*gf)[2][1] = dev_csub((*gf)[2][1], (*gf)[2][2]);
  (*gf)[2][1] = dev_cconj(dev_crealmult((*gf)[2][1], one_over_N));
  
  // calculate c2
  (*gf)[1][2] = dev_cmult(  dev_cconj(p2) ,  (*gf)[2][0]    );
  (*gf)[2][2] = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(   (*gf)[0][0]  , dev_cconj( (*gf)[1][0]) )
                     );
  (*gf)[1][2] = dev_csub((*gf)[1][2], (*gf)[2][2]);
  (*gf)[1][2] = dev_cconj(dev_crealmult((*gf)[1][2], one_over_N));
  
  // use p2 as help variable after the first step
  // calculate c3
  (*gf)[2][2] = dev_cmult(  dev_cconj(p2) ,   (*gf)[1][0]    );
  p2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(   (*gf)[0][0]  , dev_cconj((*gf)[2][0] ) )
                     );
  (*gf)[2][2] = dev_cadd((*gf)[2][2], p2);
  (*gf)[2][2] = dev_cconj(dev_crealmult((*gf)[2][2], -one_over_N));

}




//***********************  reconstruct trafo ************************************

// reconstruction of the link fields from two rows of the su3 matrix
// numbers are fetched from texture cache
__device__ void dev_reconstructtrafo_2vtexref (dev_su3_2v * field, int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  double2 gfin;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos);
    gfin = tex1Dfetch_trafo(6*pos);
  #else
    gfin = field[6*pos];
  #endif
  //first row
  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+1);
    gfin = tex1Dfetch_trafo(6*pos+1);
  #else
    gfin = field[6*pos+1];
  #endif  
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,3*pos+2);
    gfin = tex1Dfetch_trafo(6*pos+2);
  #else
    gfin = field[6*pos+2];
  #endif

  (*gf)[0][2].re = gfin.x;
  (*gf)[0][2].im = gfin.y;
  //second row
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+3);
    gfin = tex1Dfetch_trafo(6*pos+3);
  #else
    gfin = field[6*pos+3];
  #endif  
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+4);
    gfin = tex1Dfetch_trafo(6*pos+4);
  #else
    gfin = field[6*pos+4];
  #endif

  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+5);
    gfin = tex1Dfetch_trafo(6*pos+5);
  #else
    gfin = field[6*pos+5];
  #endif  
  (*gf)[1][2].re = gfin.x;
  (*gf)[1][2].im = gfin.y;
  
  //third row from cconj(cross product of first and second row)
  help1 = dev_cmult((*gf)[0][1],(*gf)[1][2]);
  help2 = dev_cmult((*gf)[0][2],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][0] = help1;

  
  help1 = dev_cmult((*gf)[0][2],(*gf)[1][0]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[1][2]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][1] = help1;
  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[0][1],(*gf)[1][0]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][2] = help1;
  
  return;
}




// su3 - dagger reconstruction from two rows  
__device__ void dev_reconstructtrafo_2vtexref_dagger (dev_su3_2v * field, int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  double2 gfin;
  
  
  //first column (minus in im for complex conj.)
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos);
    gfin = tex1Dfetch_trafo(6*pos);
  #else
    gfin = field[6*pos];
  #endif

  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+1);
    gfin = tex1Dfetch_trafo(6*pos+1);
  #else
    gfin = field[6*pos+1];
  #endif
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = -gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+2);
    gfin = tex1Dfetch_trafo(6*pos+2);
  #else
    gfin = field[6*pos+2];
  #endif
  
  (*gf)[2][0].re = gfin.x;
  (*gf)[2][0].im = -gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+3);
    gfin = tex1Dfetch_trafo(6*pos+3);
  #else
    gfin = field[6*pos+3];
  #endif  
  //second  column (minus in im for complex conj.)
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = -gfin.y;

  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+4);
    gfin = tex1Dfetch_trafo(6*pos+4);
  #else
    gfin = field[6*pos+4];
  #endif
  
  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,6*pos+5);
    gfin = tex1Dfetch_trafo(6*pos+5);
  #else
    gfin = field[6*pos+5];
  #endif  
  (*gf)[2][1].re = gfin.x;
  (*gf)[2][1].im = -gfin.y;
  
  //third column from (cross product of cconj(first column) and cconj(second column))
 
  help1 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[2][1]));
  help2 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[1][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[0][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[0][1]));
  help2 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[2][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[1][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[1][1]));
  help2 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[2][2] = help1;
  
  
  /* does this also work?
  help1 = dev_cmult((*gf)[1][0],(*gf)[2][1]);
  help2 = dev_cmult((*gf)[2][0],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[0][2] = help1;

  
  help1 = dev_cmult((*gf)[2][0],(*gf)[0][1]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[2][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[1][2] = help1;

  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][2] = help1;
  */
}









// reconstruction of the gf using 8 real parameters as 
// described in the appendix of hep-lat 0911.3191 (M.Clark et al.)
// optimized once
__device__ void dev_reconstructtrafo_8texref (dev_su3_8 * field, int pos, dev_su3* gf){

  double2 gfin, gfin2;
  double one_over_N, help;
  dev_complex p1,p2;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,4*pos);
    gfin = tex1Dfetch_trafo(4*pos);
  #else
    gfin = field[4*pos];
  #endif
  // read a2 a3
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;
  
  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(trafo_tex,4*pos+1);
    gfin2 = tex1Dfetch_trafo(4*pos+1);
  #else
    gfin2 = field[4*pos+1];
  #endif  
  (*gf)[0][2].re = gfin2.x;
  (*gf)[0][2].im = gfin2.y;  
 
  p1.re = gfin.x*gfin.x + gfin.y*gfin.y + gfin2.x*gfin2.x + gfin2.y*gfin2.y; // use later on
  one_over_N = rsqrt(p1.re); //reciprocal sqrt

  // read theta_a1, theta_c1, b1
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,4*pos+2);
    gfin = tex1Dfetch_trafo(4*pos+2);
  #else
    gfin = field[4*pos+2];
  #endif

  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(trafo_tex,4*pos+3);
    gfin2 = tex1Dfetch_trafo(4*pos+3);
  #else
    gfin2 = field[4*pos+3];
  #endif    
  // reconstruct a1 use sqrt instead of sin
  help = 1.0 - p1.re;
  if(help > 0.0){
     p1.re = sqrtf(help);
  }
  else{
    p1.re = 0.0;
  }

  
  sincos(gfin.x, &(*gf)[0][0].im, &(*gf)[0][0].re);
  (*gf)[0][0].re = (*gf)[0][0].re * p1.re;
  (*gf)[0][0].im = (*gf)[0][0].im * p1.re;
  
    
  // assign b1
  (*gf)[1][0].re = gfin2.x;
  (*gf)[1][0].im = gfin2.y;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[1][0].re;
  p2.im = one_over_N*(*gf)[1][0].im;  


  // reconstruct c1 use sqrt instead of sin
  help =1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[1][0].re * (*gf)[1][0].re - (*gf)[1][0].im * (*gf)[1][0].im;
  if(help > 0.0){
    p1.re = sqrtf(help);
  }   
  else{      
    p1.re = 0.0;  
  }
  sincos(gfin.y, &(*gf)[2][0].im, &(*gf)[2][0].re);
  (*gf)[2][0].re = (*gf)[2][0].re * p1.re; 
  (*gf)[2][0].im = (*gf)[2][0].im * p1.re;
   
  
  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[2][0].re;
  p1.im = - one_over_N*(*gf)[2][0].im;
  
 
  //use the last reconstructed gf component gf[2][2] (c3) as a help variable for b2,b3 and c2
  //this is in order to save registers and to prevent extra loading and storing from global mem
  // calculate b2
  
  (*gf)[1][1].re = p1.re*(*gf)[0][2].re;
  (*gf)[1][1].re += p1.im*(*gf)[0][2].im;
  (*gf)[1][1].im = p1.im*(*gf)[0][2].re;
  (*gf)[1][1].im -= p1.re*(*gf)[0][2].im;
  
  (*gf)[2][2].re = (*gf)[0][0].re * (*gf)[0][1].re;
  (*gf)[2][2].re += (*gf)[0][0].im * (*gf)[0][1].im;
  
  (*gf)[2][2].im = (*gf)[0][0].re * (*gf)[0][1].im;
  (*gf)[2][2].im -= (*gf)[0][0].im * (*gf)[0][1].re;
  (*gf)[2][2] = dev_cmult(p2, (*gf)[2][2]);
  
  (*gf)[1][1].re = -one_over_N*( (*gf)[1][1].re + (*gf)[2][2].re);
  (*gf)[1][1].im = -one_over_N*((*gf)[1][1].im + (*gf)[2][2].im);
  

  // calculate b3
  (*gf)[1][2].re = p1.re*(*gf)[0][1].re;
  (*gf)[1][2].re += p1.im*(*gf)[0][1].im;
  (*gf)[1][2].im = p1.im*(*gf)[0][1].re;
  (*gf)[1][2].im -= p1.re*(*gf)[0][1].im;
  
  (*gf)[2][2].re = (*gf)[0][0].re*(*gf)[0][2].re;
  (*gf)[2][2].re += (*gf)[0][0].im*(*gf)[0][2].im;
  (*gf)[2][2].im = (*gf)[0][0].re*(*gf)[0][2].im;
  (*gf)[2][2].im -= (*gf)[0][0].im*(*gf)[0][2].re;
  (*gf)[2][2] = dev_cmult(p2,(*gf)[2][2]);
  
  (*gf)[1][2].re = one_over_N*( (*gf)[1][2].re - (*gf)[2][2].re);
  (*gf)[1][2].im = one_over_N*( (*gf)[1][2].im - (*gf)[2][2].im);
  
  
  // calculate c2
  (*gf)[2][1].re = p2.re*(*gf)[0][2].re;
  (*gf)[2][1].re -= p2.im*(*gf)[0][2].im;
  (*gf)[2][1].im = -p2.re*(*gf)[0][2].im;
  (*gf)[2][1].im -= p2.im*(*gf)[0][2].re;
  
  

  (*gf)[2][2].re = (*gf)[0][0].re*(*gf)[0][1].re;
  (*gf)[2][2].re += (*gf)[0][0].im*(*gf)[0][1].im;
  (*gf)[2][2].im = (*gf)[0][0].re* (*gf)[0][1].im;
  (*gf)[2][2].im -= (*gf)[0][0].im* (*gf)[0][1].re;
  help = (*gf)[2][2].re;
  (*gf)[2][2].re = p1.re*(*gf)[2][2].re;
  (*gf)[2][2].re += p1.im*(*gf)[2][2].im;
  (*gf)[2][2].im = p1.re*(*gf)[2][2].im - p1.im*help;
  
  
  (*gf)[2][1].re = one_over_N*((*gf)[2][1].re - (*gf)[2][2].re);
  (*gf)[2][1].im = one_over_N*((*gf)[2][1].im - (*gf)[2][2].im);
  
  // now we have to use p2 and p1 as a help variable, as this is not 
  // needed any more after the first
  // step
  // calculate c3
  (*gf)[2][2].re = p2.re * (*gf)[0][1].re;
  (*gf)[2][2].re -= p2.im * (*gf)[0][1].im;
  (*gf)[2][2].im = - p2.im*(*gf)[0][1].re;
  (*gf)[2][2].im -= p2.re*(*gf)[0][1].im;
  
  p2.re = (*gf)[0][0].re * (*gf)[0][2].re;
  p2.re += (*gf)[0][0].im * (*gf)[0][2].im;
  p2.im = (*gf)[0][0].re * (*gf)[0][2].im;
  p2.im -= (*gf)[0][0].im * (*gf)[0][2].re;
  p2 = dev_cmult(  dev_cconj(p1) , p2);
  
  (*gf)[2][2] = dev_cadd((*gf)[2][2], p2);
  (*gf)[2][2] = dev_crealmult((*gf)[2][2], -one_over_N);
}





__device__ void dev_reconstructtrafo_8texref_dagger (dev_su3_8 * field, int pos, dev_su3* gf){
  double2 gfin, gfin2;
  double one_over_N, help;
  dev_complex p1,p2;
  
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,4*pos);
    gfin = tex1Dfetch_trafo(4*pos);
  #else
    gfin = field[4*pos];
  #endif  

  // read a2 a3
  (*gf)[1][0].re = gfin.x;
  (*gf)[1][0].im = -gfin.y;
  
  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(trafo_tex,4*pos+1);
    gfin2 = tex1Dfetch_trafo(4*pos+1);
  #else
    gfin2 = field[4*pos+1];
  #endif    
  
  (*gf)[2][0].re = gfin2.x;
  (*gf)[2][0].im = -gfin2.y;  
 
  p1.re = gfin.x*gfin.x + gfin.y*gfin.y + gfin2.x*gfin2.x + gfin2.y*gfin2.y; // use later on
  one_over_N = rsqrt(p1.re);  // reciprocal sqrt

  
  // read theta_a1, theta_c1, b1
  #ifdef USETEXTURE
    //gfin = tex1Dfetch(trafo_tex,4*pos+2);
    gfin = tex1Dfetch_trafo(4*pos+2);
  #else
    gfin = field[4*pos+2];
  #endif


  #ifdef USETEXTURE
    //gfin2 = tex1Dfetch(trafo_tex,4*pos+3);
    gfin2 = tex1Dfetch_trafo(4*pos+3);
  #else
    gfin2 = field[4*pos+3];
  #endif
  
  // reconstruct a1
  help = 1.0 - p1.re;
  if(help > 0.0){
     p1.re = sqrtf(help);   
  }
  else{
    p1.re = 0.0;
  }
  //(*gf)[0][0].re = p1.re*cos(gfin.x);
  //(*gf)[0][0].im = -p1.re*sin(gfin.x);
  
  sincos(gfin.x, &(*gf)[0][0].im, &(*gf)[0][0].re);
  (*gf)[0][0].re = (*gf)[0][0].re * p1.re;
  (*gf)[0][0].im = -(*gf)[0][0].im * p1.re;
    
  
  // assign b1
  (*gf)[0][1].re = gfin2.x;
  (*gf)[0][1].im = -gfin2.y;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[0][1].re;
  p2.im = -one_over_N*(*gf)[0][1].im;  


  // reconstruct c1
  help = 1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[0][1].re * (*gf)[0][1].re - (*gf)[0][1].im * (*gf)[0][1].im;
  if(help > 0.0){
    p1.re = sqrtf(help);
  }
  else{
    p1.re = 0.0;
  }
  //(*gf)[0][2].re = p1.re*cos(gfin.y);
  //(*gf)[0][2].im = -p1.re*sin(gfin.y);
  
  sincos(gfin.y, &(*gf)[0][2].im, &(*gf)[0][2].re);
  (*gf)[0][2].re = (*gf)[0][2].re * p1.re;
  (*gf)[0][2].im = -(*gf)[0][2].im * p1.re;
     
  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[0][2].re;
  p1.im = one_over_N*(*gf)[0][2].im;
  
  //use the last reconstructed gf component gf[2][2] (c3) as a help variable for b2,b3 and c2
  //this is in order to save registers and to prevent extra loading and storing from global mem
  // calculate b2
  (*gf)[1][1] = dev_cmult(p1,   (*gf)[2][0]    );
  (*gf)[2][2] = dev_cmult(p2, dev_cmult( (*gf)[0][0] , dev_cconj((*gf)[1][0] ))  );
  (*gf)[1][1] = dev_cadd((*gf)[1][1], (*gf)[2][2]);
  (*gf)[1][1] = dev_cconj(dev_crealmult((*gf)[1][1], -one_over_N));
  
  // calculate b3
  (*gf)[2][1] = dev_cmult(p1,   (*gf)[1][0]    );
  (*gf)[2][2] = dev_cmult(p2, dev_cmult( (*gf)[0][0] , dev_cconj((*gf)[2][0] ))  );
  (*gf)[2][1] = dev_csub((*gf)[2][1], (*gf)[2][2]);
  (*gf)[2][1] = dev_cconj(dev_crealmult((*gf)[2][1], one_over_N));
  
  // calculate c2
  (*gf)[1][2] = dev_cmult(  dev_cconj(p2) ,  (*gf)[2][0]    );
  (*gf)[2][2] = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(   (*gf)[0][0]  , dev_cconj( (*gf)[1][0]) )
                     );
  (*gf)[1][2] = dev_csub((*gf)[1][2], (*gf)[2][2]);
  (*gf)[1][2] = dev_cconj(dev_crealmult((*gf)[1][2], one_over_N));
  
  // use p2 as help variable after the first step
  // calculate c3
  (*gf)[2][2] = dev_cmult(  dev_cconj(p2) ,   (*gf)[1][0]    );
  p2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(   (*gf)[0][0]  , dev_cconj((*gf)[2][0] ) )
                     );
  (*gf)[2][2] = dev_cadd((*gf)[2][2], p2);
  (*gf)[2][2] = dev_cconj(dev_crealmult((*gf)[2][2], -one_over_N));


}






void show_su3_2v(dev_su3_2v * M){
 complex a0, a1, a2;
 complex b0, b1, b2;
 complex c0, c1, c2;
 complex help1, help2;
    
 printf("(%e,%e) ", (*M).x, (*M).y);
 printf("(%e,%e) ", (*(M+1)).x, (*(M+1)).y);
 printf("(%e,%e) ", (*(M+2)).x, (*(M+2)).y);
 printf("\n");
 
 printf("(%e,%e) ", (*(M+3)).x, (*(M+3)).y);
 printf("(%e,%e) ", (*(M+4)).x, (*(M+4)).y);
 printf("(%e,%e) ", (*(M+5)).x, (*(M+5)).y);
 
 printf("\n");
 double re1 = (*M).x * (*(M+3)).x + (*M).y * (*(M+3)).y;
 double im1 = - (*M).x * (*(M+3)).y + (*M).y * (*(M+3)).x;

 double re2 = (*(M+1)).x * (*(M+4)).x + (*(M+1)).y * (*(M+4)).y;
 double im2 = - (*(M+1)).x * (*(M+4)).y + (*(M+1)).y * (*(M+4)).x;
 
 double re3 = (*(M+2)).x * (*(M+5)).x + (*(M+2)).y * (*(M+5)).y;
 double im3 = - (*(M+2)).x * (*(M+5)).y + (*(M+2)).y * (*(M+5)).x;
 
 double allre = (re1+re2+re3);
 double allim = im1+im2+im3;
 
 printf("a b* = (%.16e, %.16e) \n", allre, allim);
 
 double norm = (*M).x*(*M).x + (*M).y*(*M).y + (*(M+1)).x*(*(M+1)).x + (*(M+1)).y*(*(M+1)).y + (*(M+2)).x*(*(M+2)).x + (*(M+2)).y*(*(M+2)).y;
 printf("a^2 = %.16e\n", norm);
 
 norm = (*(M+3)).x * (*(M+3)).x + (*(M+3)).y * (*(M+3)).y + (*(M+4)).x * (*(M+4)).x + (*(M+4)).y * (*(M+4)).y + (*(M+5)).x * (*(M+5)).x + (*(M+5)).y * (*(M+5)).y; 
 printf("b^2 = %.16e\n", norm);
 
 a0.re = (*M).x;
 a0.im = (*M).y;
 a1.re = (*(M+1)).x;
 a1.im = (*(M+1)).y; 
 a2.re = (*(M+2)).x;
 a2.im = (*(M+2)).y; 
 
 b0.re = (*(M+3)).x;
 b0.im = (*(M+3)).y;
 b1.re = (*(M+4)).x;
 b1.im = (*(M+4)).y; 
 b2.re = (*(M+5)).x;
 b2.im = (*(M+5)).y;  
 
 
  // c = (a X b)*
  help1 = cmult(a1,b2);
  help2 = cmult(a2,b1);
  help1 = cconj(csub(help1,help2));
  c0 = help1;

  
  help1 = cmult(a2,b0);
  help2 = cmult(a0,b2);
  help1 = cconj(csub(help1,help2));
  c1 = help1;
  
  help1 = cmult(a0,b1);
  help2 = cmult(a1,b0);
  help1 = cconj(csub(help1,help2));
  c2 = help1;
  
  
  norm = c0.re*c0.re + c0.im*c0.im + c1.re*c1.re + c1.im*c1.im + c2.re*c2.re + c2.im*c2.im;
 printf("c^2 = %.16e\n", norm); 
  
  
}






__device__ void dev_su3zero(dev_su3* M){
 int i,j;
 dev_complex czero = dev_initcomplex(0.0,0.0);
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){
      (*M)[i][j] = czero;
    }
  }
return;
}



// erg = M^+
__device__ void dev_su3dagger(dev_su3 * erg, dev_su3 * M){
  int i,j;
  
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){  
      (*erg)[i][j] = dev_cconj((*M)[j][i]);
    }
  }
return;
}


__device__ dev_complex dev_su3trace(dev_su3 * M){
  dev_complex erg;
  int i;
  erg = dev_initcomplex(0.0,0.0);
  for(i=0; i<3; i++){
    erg = dev_cadd(erg, (*M)[i][i]);
  }
return erg;
}



__device__ double dev_su3Retrace(dev_su3 * M){
  double erg;
  int i;
  erg = 0.0;
  for(i=0; i<3; i++){
    erg = erg + (*M)[i][i].re;
  }
return erg;
}



//erg = C*M, C complex 
__device__ void dev_su3skalarmult(dev_su3 * erg, dev_complex skalar, dev_su3 * M){
  int i,j;
  
  for(i=0;i<3;i++){
    for(j=0;j<3;j++){
      (*erg)[i][j] = dev_cmult(skalar,(*M)[i][j]);
    }
  }

return;
}


__device__ void dev_su2_ti_su2(dev_su2 * r, dev_su2* a, dev_su2* b){

  (*r).a.x =   (*a).a.x * (*b).a.x - (*a).a.y * (*b).a.y 
             - (*a).b.x * (*b).b.x - (*a).b.y * (*b).b.y;
  
  (*r).a.y =   (*a).a.x * (*b).a.y + (*a).a.y * (*b).a.x 
             - (*a).b.x * (*b).b.y + (*a).b.y * (*b).b.x;
  
  (*r).b.x =   (*a).a.x * (*b).b.x + (*a).b.x * (*b).a.x 
             - (*a).b.y * (*b).a.y + (*a).a.y * (*b).b.y;
  
  (*r).b.y =   (*a).a.x * (*b).b.y + (*a).b.y * (*b).a.x 
             - (*a).a.y * (*b).b.x + (*a).b.x * (*b).a.y;

}




// to := from
__device__ void dev_su3copy( dev_su3 * to, dev_su3 * from){
  int i,j;
  for(i=0; i<3; i++){
    for(j=0; j<3; j++){
      (*to)[i][j] = (*from)[i][j];
    }
  }
return;
}



// u = v * w
__device__ void dev_su3_ti_su3(dev_su3* u, dev_su3 * v, dev_su3 * w){
  dev_complex help1, help2;
  dev_complex zero = dev_initcomplex(0.0,0.0);
  int i,j,k;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){
    
      help2 = zero;
      for(k=0; k<3; k++){
          help1 = dev_cmult((*v)[i][k],(*w)[k][j]);
          help2 = dev_cadd(help1, help2);
        }
        (*u)[i][j] = help2;    
    }
  }
}





// u = u + v * w
__device__ void dev_add_su3_ti_su3(dev_su3* u, dev_su3 * v, dev_su3 * w){
  dev_complex help1, help2;
  dev_complex zero = dev_initcomplex(0.0,0.0);
  int i,j,k;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){
    
      help2 = zero;
      for(k=0; k<3; k++){
          help1 = dev_cmult((*v)[i][k],(*w)[k][j]);
          help2 = dev_cadd(help1, help2);
        }
        (*u)[i][j].re += help2.re;  
        (*u)[i][j].im += help2.im; 
    }
  }
}



// u = v* w^+
__device__ void dev_su3_ti_su3d(dev_su3* u, dev_su3 * v, dev_su3 * w){
  dev_complex help1, help2;
  dev_complex zero = dev_initcomplex(0.0,0.0);
  int i,j,k;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      help2 = zero;
      for(k=0; k<3; k++){
          help1 = dev_cmult((*v)[i][k],dev_cconj( (*w)[j][k] ) );
          help2 = dev_cadd(help1, help2);
        }
        (*u)[i][j] = help2;   
    }
  }
}


// a = a-b
__device__ void dev_su3_sub(dev_su3* a, dev_su3* b){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*a)[i][j] = dev_csub((*a)[i][j], (*b)[i][j]);
    }
  }
}





// c = a-b
__device__ void dev_su3_sub_assign(dev_su3* c,dev_su3* a, dev_su3* b){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*c)[i][j] = dev_csub((*a)[i][j], (*b)[i][j]);
    }
  }
}





// a = a+b
__device__ void dev_su3_add(dev_su3* a, dev_su3* b){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*a)[i][j] = dev_cadd((*a)[i][j], (*b)[i][j]);
    }
  }
}


// c = a+b
__device__ void dev_su3_add_assign(dev_su3* c, dev_su3* a, dev_su3* b){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*c)[i][j] = dev_cadd((*a)[i][j], (*b)[i][j]);
    }
  }
}



// a:=R*a
__device__ void dev_su3_real_mult(dev_su3* a, double R){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*a)[i][j].re = (*a)[i][j].re*R;
      (*a)[i][j].im = (*a)[i][j].im*R;
    }
  }
}




// erg=R*a
__device__ void dev_su3_real_mult_assign(dev_su3* erg, dev_su3* a, double R){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*erg)[i][j].re = (*a)[i][j].re*R;
      (*erg)[i][j].im = (*a)[i][j].im*R;
    }
  }
}




// a:= b
__device__ void dev_su3_assign(dev_su3* a, dev_su3* b){
  int i,j;
  for(i=0; i<3;i++){
    for(j=0; j<3; j++){ 
      (*a)[i][j] = (*b)[i][j];
    }
  }
}



__device__ void dev_su3_normalize(dev_su3* u){
  int i;
  double len;
  dev_complex proj, help1, help2;
  //length of first row (a)
  len = ((*u)[0][0].re*(*u)[0][0].re + (*u)[0][0].im*(*u)[0][0].im) +
        ((*u)[0][1].re*(*u)[0][1].re + (*u)[0][1].im*(*u)[0][1].im) +
        ((*u)[0][2].re*(*u)[0][2].re + (*u)[0][2].im*(*u)[0][2].im) ;
  //normalize a
  
  len =  rsqrt(len); // 1/sqrt(len)
  
  #pragma unroll 3
  for(i=0; i<3; i++){
    (*u)[0][i].re = (*u)[0][i].re*len;
    (*u)[0][i].im = (*u)[0][i].im*len;
  }     
  
  //projection on second row b (f=a*b)
  proj = dev_initcomplex(0.0,0.0);
  #pragma unroll 3
  for(i=0; i<3; i++){
    proj = dev_cadd( proj , dev_cmult(  (*u)[1][i] ,  dev_cconj( (*u)[0][i])   ));
  }
  
  
  //orthogonalize -> new b
  #pragma unroll 3
  for(i=0; i<3; i++){
    (*u)[1][i] = dev_csub( (*u)[1][i] , dev_cmult(proj, (*u)[0][i]  )  );
  }
  
  
 
  // get length b
  len = ((*u)[1][0].re*(*u)[1][0].re + (*u)[1][0].im*(*u)[1][0].im) +
        ((*u)[1][1].re*(*u)[1][1].re + (*u)[1][1].im*(*u)[1][1].im) +
        ((*u)[1][2].re*(*u)[1][2].re + (*u)[1][2].im*(*u)[1][2].im) ;
  
  len =  rsqrt(len); // 1/sqrt(len)
  // normalize b
  #pragma unroll 3
  for(i=0; i<3; i++){
    (*u)[1][i].re = (*u)[1][i].re*len;
    (*u)[1][i].im = (*u)[1][i].im*len;
  }


  // c = (a X b)*
  help1 = dev_cmult((*u)[0][1],(*u)[1][2]);
  help2 = dev_cmult((*u)[0][2],(*u)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*u)[2][0] = help1;

  
  help1 = dev_cmult((*u)[0][2],(*u)[1][0]);
  help2 = dev_cmult((*u)[0][0],(*u)[1][2]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*u)[2][1] = help1;
  
  help1 = dev_cmult((*u)[0][0],(*u)[1][1]);
  help2 = dev_cmult((*u)[0][1],(*u)[1][0]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*u)[2][2] = help1;
        
}









extern "C" int bind_texture_gf(dev_su3_2v * gfield){
 //printf("Binding texture to gaugefield\n");
 
 #ifdef GF_8
 size_t size = sizeof(double2)*4*VOLUME*4;
 #else
 size_t size = sizeof(double2)*6*VOLUME*4;
 #endif
 
 hipGetTextureReference(&gf_texRefPtr, HIP_SYMBOL("gf_tex"));
 gf_channelDesc =  hipCreateChannelDesc<int4>();
 hipBindTexture(0, gf_texRefPtr, (int4 *) gfield, &gf_channelDesc, size);
 //printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}


extern "C" int unbind_texture_gf(){
 //printf("Unbinding texture from gaugefield\n");
 hipUnbindTexture(gf_texRefPtr);
 //printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}




extern "C" int bind_texture_trafo(dev_su3_2v * trafofield){
 //printf("Binding texture to trafo field\n");
 
 #ifdef GF_8
 size_t size = sizeof(double2)*4*VOLUME;
 #else
 size_t size = sizeof(double2)*6*VOLUME;
 #endif
 
 hipGetTextureReference(&trafo_texRefPtr, HIP_SYMBOL("trafo_tex"));
 trafo_channelDesc =  hipCreateChannelDesc<int4>();
 hipBindTexture(0, trafo_texRefPtr, (int4 *) trafofield, &trafo_channelDesc, size);
 //printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}


extern "C" int unbind_texture_trafo(){
 //printf("Unbinding texture from trafo field\n");
 hipUnbindTexture(trafo_texRefPtr);
 //printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}



