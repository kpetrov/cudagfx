#include "hip/hip_runtime.h"

#define MAIN_PROGRAM

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include "cudaglobal.h"
#include <math.h>
#include "global.h"
#include <getopt.h>
#include <time.h>
#include <assert.h>
#include "dev_su3.h"

extern "C" {
 #include "complex.h"
 #include "gauge_io.h"
 #include "rngs.h"
 #include "su3manip.h"
 #include "observables.h"
 #include "read_input.h"
}



#  define CUDA_SAFE_CALL( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error : %s.\n",  hipGetErrorString( err) );   \
        exit(EXIT_FAILURE);                                                  \
    } }








__device__  int  dev_LX,dev_LY,dev_LZ,dev_T,dev_VOLUME;

// the reduction fields for the global sums of the functional F, dAdA and maxdAdA
__device__ double * dev_redfield_F;
__device__ double * dev_redfield_dAdA;
__device__ double * dev_redfield_maxdAdA;
__device__ double * dev_redfield_plaq;
__constant__ __device__ double sa_beta;
__constant__ __device__ double therm_beta;

/* include the cuda code files here, necessary, because nvcc does not support external calls, all cuda functions must be inlined*/

#include "dev_su3.cu"
#include "overrelax.cu"
#include "MersenneTwister.cu"
#include "simulated_annealing.cu"
#include "heatbath_thermalization.cu"





extern int read_gf_ildg(su3 gf[], char* filename);



void initnn(){
  int t,x,y,z,pos, count;
  count=0;
  for(t=0;t<T;t++){
    for(z=0; z<LZ; z++){
      for(y=0; y<LY; y++){
        for(x=0; x<LX; x++){
          pos= x + LX*(y + LY*(z + LZ*t));
          ind[count] = pos;
          //plus direction
          nn[8*pos+3] = x + LX*(y + LY*(z + LZ*((t+1)%T)));
          nn[8*pos+2] = x + LX*(y + LY*((z+1)%LZ + LZ*t));
          nn[8*pos+1] = x + LX*((y+1)%LY + LY*(z + LZ*t));
          nn[8*pos+0] = (x+1)%LX + LX*(y + LY*(z + LZ*t));
          //minus direction
          if(t==0){
            nn[8*pos+7] = x + LX*(y + LY*(z + LZ*((T-1))));
          }
          else{
            nn[8*pos+7] = x + LX*(y + LY*(z + LZ*((t-1))));
          }
          if(z==0){
            nn[8*pos+6] = x + LX*(y + LY*((LZ-1) + LZ*t));
          }
          else{
            nn[8*pos+6] = x + LX*(y + LY*((z-1) + LZ*t));
          }
          if(y==0){
            nn[8*pos+5] = x + LX*((LY-1) + LY*(z + LZ*t));
          }
          else{
            nn[8*pos+5] = x + LX*((y-1) + LY*(z + LZ*t));
          }
          if(x==0){
            nn[8*pos+4] = (LX-1) + LX*(y + LY*(z + LZ*t));
          }
          else{
            nn[8*pos+4] = (x-1) + LX*(y + LY*(z + LZ*t));
          }          
        
        
        count++;
        }
      }
    } 
  }
}








//initialize nearest-neighbour table for gpu with even-odd enabled
//init_nn must have been called before for initialization of nn
void initnn_eo(){
  int x,y,z,t,index,nnpos,j, count;
  int evenpos=0;
  int oddpos=0;

  // here we initialize the conversion field lexic2eo
  evenpos=0;
  oddpos=0;
  count=0;
  for(t=0;t<T;t++){
    for(z=0;z<LZ;z++){
      for(y=0;y<LY;y++){
        for(x=0;x<LX;x++){
          if(  ((x+y+z+t) %2)==0){
            lexic2eo[count] = evenpos;
            evenpos++;
          }
          else{
            lexic2eo[count] = oddpos;
            oddpos++;
          }
          count++;
        }
      }
    }
  }
  
  
  
  
  evenpos=0;
  oddpos=0;
  count=0;
  for(t=0;t<T;t++){
   for(z=0;z<LZ;z++){
    for(y=0;y<LY;y++){
     for(x=0;x<LX;x++){
          index = ind[count];
          
          if(((t+x+y+z)%2 == 0)){
            nnpos = lexic2eo[index];
            for(j=0;j<4;j++){
              nn_eo[8*nnpos+j] = lexic2eo[ nn[8*index+j] ];
            }
            for(j=0;j<4;j++){
              nn_eo[8*nnpos+4+j] = lexic2eo[ nn[8*index+4+j] ];
            }
            eoidx_even[evenpos] = index;
            evenpos++;
          }
          else{
            nnpos = lexic2eo[index];
            for(j=0;j<4;j++){
              nn_oe[8*nnpos+j] = lexic2eo[ nn[8*index+j] ];
            }
            for(j=0;j<4;j++){
              nn_oe[8*nnpos+4+j] = lexic2eo[ nn[8*index+4+j] ];
            }
            eoidx_odd[oddpos] = index;
            oddpos++;
          }
        
        count++;
        }
      }
    }
  }
}







void init_gaugefixing(su3* gf, su3* trafo){
hipError_t cudaerr;

  // the gauge field
  #ifdef GF_8
  /* allocate 8 doubles of gf = 4*4*VOLUME double2's*/
  size_t dev_gfsize = 4*4*VOLUME * sizeof(dev_su3_8);
  #else
  /* allocate 2 rows of gf = 6*4*VOLUME double2's*/
  size_t dev_gfsize = 6*4*VOLUME * sizeof(dev_su3_2v); 
  #endif
  
  if((cudaerr=hipMalloc((void **) &dev_gf, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field on device\n");
  }  

 #ifdef USETEXTURE
  /*
  if((cudaerr=hipMalloc((void **) &dev_gf2, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field 2 on device\n");
  } 
  */
 #endif
  
  #ifdef GF_8
  h2d_gf = (dev_su3_8 *)malloc(dev_gfsize); // Allocate conversion gf on host
  su3to8(gf,h2d_gf);  
  #else
  h2d_gf = (dev_su3_2v *)malloc(dev_gfsize); // Allocate conversion gf on host
  su3to2v(gf,h2d_gf);
  #endif
  hipMemcpy(dev_gf, h2d_gf, dev_gfsize, hipMemcpyHostToDevice);
  



// the trafo fields
  #ifdef GF_8
  /* allocate 8 doubles of trafo = 4*VOLUME double2's*/
  dev_gfsize = 4*VOLUME * sizeof(dev_su3_8);
  #else
  /* allocate 2 rows of gf = 6*VOLUME double2's*/
  dev_gfsize = 6*VOLUME * sizeof(dev_su3_2v); 
  #endif
  
  if((cudaerr=hipMalloc((void **) &dev_trafo1, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of trafo field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated trafo field 1 on device\n");
  }  
  
  #ifdef USETEXTURE //we only need a second trafo field, if we use textures as texture fields are read-only!
  if((cudaerr=hipMalloc((void **) &dev_trafo2, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of trafo field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated trafo field 2 on device\n");
  }
  #endif
  
  #ifdef GF_8
  h2d_trafo = (dev_su3_8 *)malloc(dev_gfsize); 
  su3to8_trafo(trafo,h2d_trafo);  
  #else
  h2d_trafo = (dev_su3_2v *)malloc(dev_gfsize); 
  su3to2v_trafo(trafo,h2d_trafo);
  #endif
  hipMemcpy(dev_trafo1, h2d_trafo, dev_gfsize, hipMemcpyHostToDevice);
  
  #ifdef USETEXTURE
  hipMemcpy(dev_trafo2, h2d_trafo, dev_gfsize, hipMemcpyHostToDevice);
  #endif
  

//grid 
  size_t nnsize = 8*VOLUME*sizeof(int);
  nn = (int *) malloc(nnsize);
  hipMalloc((void **) &dev_nn, nnsize);
  
  size_t indsize = VOLUME*sizeof(int);
  ind = (int *) malloc(indsize);
  
  lexic2eo = (int *) malloc(indsize);
 
 
  // nearest neighbours EO
  size_t nnsize_evenodd = (size_t)8*VOLUME/2*sizeof(int);
  nn_oe = (int *) malloc(nnsize_evenodd);
  hipMalloc((void **) &dev_nn_oe, nnsize_evenodd);
  nn_eo = (int *) malloc(nnsize_evenodd);
  hipMalloc((void **) &dev_nn_eo, nnsize_evenodd);
  
  // index EO
  size_t indsize_evenodd = (size_t)VOLUME/2*sizeof(int);
  eoidx_even = (int *) malloc(indsize_evenodd);
  hipMalloc((void **) &dev_eoidx_even, indsize_evenodd);
  eoidx_odd = (int *) malloc(indsize_evenodd);
  hipMalloc((void **) &dev_eoidx_odd, indsize_evenodd);
  
  
  initnn();
  initnn_eo();
  
  //shownn();
  //showcompare_gf(T-1, LX-1, LY-1, LZ-1, 3);
  
  // copy to device index arrays
  hipMemcpy(dev_nn, nn, nnsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_eo, nn_eo, nnsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_oe, nn_oe, nnsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_even, eoidx_even, indsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_odd, eoidx_odd, indsize_evenodd, hipMemcpyHostToDevice);


  output_size = LZ*T*sizeof(double); // parallel in t and z direction
  hipMalloc((void **) &dev_output, output_size);   // output array
  double * host_output = (double*) malloc(output_size);

  int grid[5];
  grid[0]=LX; grid[1]=LY; grid[2]=LZ; grid[3]=T; grid[4]=VOLUME;
 
  hipMalloc((void **) &dev_grid, (size_t)(5*sizeof(int)));
  hipMemcpy(dev_grid, &(grid[0]), (size_t)(5*sizeof(int)), hipMemcpyHostToDevice);
  
  //init grid
    dev_gfix_init<<< 1, 1 >>> (dev_grid);
  
  
  //reduction field for functional  
  if(VOLUME%BLOCK != 0){
   printf("Error: VOLUME is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  

  int redfieldsize = VOLUME/BLOCK;
  printf("VOLUME/BLOCK = %d\n", VOLUME/BLOCK);
  hipMalloc((void **) &dev_redfield_F, redfieldsize*sizeof(double));
  if((redfield_F = (double*)malloc(redfieldsize*sizeof(double)))==(void*)NULL){
    fprintf(stderr,"Error in init_gaugefixing: malloc error(F)\n");
  }
  hipMalloc((void **) &dev_redfield_dAdA, redfieldsize*sizeof(double));
  if((redfield_dAdA = (double*)malloc(redfieldsize*sizeof(double)))==(void*)NULL){
    fprintf(stderr,"Error in init_gaugefixing: malloc error(dAdA)\n");
  }   
  
  hipMalloc((void **) &dev_redfield_maxdAdA, redfieldsize*sizeof(double));
  if((redfield_maxdAdA = (double*)malloc(redfieldsize*sizeof(double)))==(void*)NULL){
    fprintf(stderr,"Error in init_gaugefixing: malloc error(maxdAdA)\n");
  } 

  hipMalloc((void **) &dev_redfield_plaq, T*sizeof(double));
  if((redfield_plaq = (double*)malloc(T*sizeof(double)))==(void*)NULL){
    fprintf(stderr,"Error in init_gaugefixing: malloc error(plaq)\n");
  }
  
   
printf("%s\n", hipGetErrorString(hipGetLastError()));

}






void finalize_gaugefixing(){

  hipFree(dev_gf);
  hipFree(dev_trafo1);
  #ifdef USETEXTURE
  //hipFree(dev_gf2);
  hipFree(dev_trafo2);
  #endif
  
  hipFree(dev_grid);
  hipFree(dev_output);
  hipFree(dev_nn);
  hipFree(dev_redfield_F);
  hipFree(dev_redfield_dAdA);
  hipFree(dev_redfield_maxdAdA);
  hipFree(dev_redfield_plaq);
  hipFree(dev_nn_eo);
  hipFree(dev_nn_oe);
  hipFree(dev_eoidx_even);
  hipFree(dev_eoidx_odd);
  free(h2d_gf);
  free(h2d_trafo);
  free(redfield_F);
  free(redfield_dAdA);
  free(redfield_maxdAdA);
  free(redfield_plaq);
  free(nn);
  free(nn_eo);
  free(nn_oe);
  free(eoidx_even);
  free(eoidx_odd);
  free(lexic2eo);
  free(ind);
}





void init_thermalization(su3* gf){
hipError_t cudaerr;

  // the gauge field
  #ifdef GF_8
  /* allocate 8 doubles of gf = 4*4*VOLUME double2's*/
  size_t dev_gfsize = 4*4*VOLUME * sizeof(dev_su3_8);
  #else
  /* allocate 2 rows of gf = 6*4*VOLUME double2's*/
  size_t dev_gfsize = 6*4*VOLUME * sizeof(dev_su3_2v); 
  #endif
  
  if((cudaerr=hipMalloc((void **) &dev_gf, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field on device\n");
  }  


  if((cudaerr=hipMalloc((void **) &dev_gf2, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field 2 on device\n");
  } 
 


  
  #ifdef GF_8
  h2d_gf = (dev_su3_8 *)malloc(dev_gfsize); // Allocate conversion gf on host
  su3to8(gf,h2d_gf);  
  #else
  h2d_gf = (dev_su3_2v *)malloc(dev_gfsize); // Allocate conversion gf on host
  su3to2v(gf,h2d_gf);
  #endif
  hipMemcpy(dev_gf, h2d_gf, dev_gfsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_gf2, h2d_gf, dev_gfsize, hipMemcpyHostToDevice);


// the staples field; we do EVEN/ODD update, so we only need half the gauge field size
  #ifdef GF_8
  /* allocate 8 doubles of gf = 4*4*VOLUME/2 double2's*/
   dev_gfsize = 4*4*VOLUME/2 * sizeof(dev_su3_8);
  #else
  /* allocate 2 rows of gf = 6*4*VOLUME/2 double2's*/
   dev_gfsize = 6*4*VOLUME/2 * sizeof(dev_su3_2v); 
  #endif
  
  if((cudaerr=hipMalloc((void **) &dev_staples, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of staple field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated staple field on device\n");
  }  
  
  
//grid 
  size_t nnsize = 8*VOLUME*sizeof(int);
  nn = (int *) malloc(nnsize);
  hipMalloc((void **) &dev_nn, nnsize);
  
  size_t indsize = VOLUME*sizeof(int);
  ind = (int *) malloc(indsize);
  
  lexic2eo = (int *) malloc(indsize);
 
 
  // nearest neighbours EO
  size_t nnsize_evenodd = (size_t)8*VOLUME/2*sizeof(int);
  nn_oe = (int *) malloc(nnsize_evenodd);
  hipMalloc((void **) &dev_nn_oe, nnsize_evenodd);
  nn_eo = (int *) malloc(nnsize_evenodd);
  hipMalloc((void **) &dev_nn_eo, nnsize_evenodd);
  
  // index EO
  size_t indsize_evenodd = (size_t)VOLUME/2*sizeof(int);
  eoidx_even = (int *) malloc(indsize_evenodd);
  hipMalloc((void **) &dev_eoidx_even, indsize_evenodd);
  eoidx_odd = (int *) malloc(indsize_evenodd);
  hipMalloc((void **) &dev_eoidx_odd, indsize_evenodd);
  
  
  initnn();
  initnn_eo();
  
  //shownn();
  //showcompare_gf(T-1, LX-1, LY-1, LZ-1, 3);
  
  // copy to device index arrays
  hipMemcpy(dev_nn, nn, nnsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_eo, nn_eo, nnsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_oe, nn_oe, nnsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_even, eoidx_even, indsize_evenodd, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_odd, eoidx_odd, indsize_evenodd, hipMemcpyHostToDevice);


  output_size = LZ*T*sizeof(double); // parallel in t and z direction
  hipMalloc((void **) &dev_output, output_size);   // output array
  double * host_output = (double*) malloc(output_size);

  int grid[5];
  grid[0]=LX; grid[1]=LY; grid[2]=LZ; grid[3]=T; grid[4]=VOLUME;
 
  hipMalloc((void **) &dev_grid, (size_t)(5*sizeof(int)));
  hipMemcpy(dev_grid, &(grid[0]), (size_t)(5*sizeof(int)), hipMemcpyHostToDevice);
  
  //init grid
    dev_gfix_init<<< 1, 1 >>> (dev_grid);
  
  
  //reduction field for functional  
  if(VOLUME%BLOCK != 0){
   printf("Error: VOLUME is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  
  hipMalloc((void **) &dev_redfield_plaq, T*sizeof(double));
  if((redfield_plaq = (double*)malloc(T*sizeof(double)))==(void*)NULL){
    fprintf(stderr,"Error in init_gaugefixing: malloc error(plaq)\n");
  }
  
   
  printf("%s\n", hipGetErrorString(hipGetLastError()));

}






void finalize_thermalization(){
  hipFree(dev_gf);
  hipFree(dev_staples);
  hipFree(dev_grid);
  hipFree(dev_output);
  hipFree(dev_nn);
  hipFree(dev_redfield_plaq);
  hipFree(dev_nn_eo);
  hipFree(dev_nn_oe);
  hipFree(dev_eoidx_even);
  hipFree(dev_eoidx_odd);
  free(h2d_gf);
  free(redfield_plaq);
  free(nn);
  free(nn_eo);
  free(nn_oe);
  free(eoidx_even);
  free(eoidx_odd);
  free(lexic2eo);
  free(ind);
}





void intro(){
  fprintf(stdout, "\n");
  fprintf(stdout, "########       This is cuda_GF                       ########\n");
  fprintf(stdout, "########       a program to fix lattice Landau gauge ########\n");
  fprintf(stdout, "########       Copyright: Florian Burger             ########\n\n\n");
  
}


void usage() {
  fprintf(stdout, "Code to compute Landau gauge on gauge field\n");
  fprintf(stdout, "Usage:   cudagaugefix -i [inputfile] [gaugefile]\n");
  exit(0);
}


int main(int argc, char *argv[]){

  int ret;
  double F,dada;
  double plaq;
  int c;
  int gfDEVICE;
  int gridsize;  
  if(VOLUME%BLOCK != 0){
   printf("Error: VOLUME is not a multiple of BLOCK. Aborting...\n");
   exit(100);
  }
  dim3 blockdim(BLOCK,1,1);
  if( VOLUME >= BLOCK){
   gridsize =VOLUME/BLOCK;
  }
  else{
   gridsize=1;
  }
  dim3 griddim(gridsize,1,1);   
  
  
  char inputfilename[100];
  char gaugefilename[100];
  char fixedgaugename[100];

if ((argc != 4) && (argc != 3)){
    // usage();
} 

intro();
gfDEVICE=0;
while ((c = getopt(argc, argv, "h?:i:d:f:")) != -1) {
      switch (c) {
          case 'i':
              strcpy ( &(inputfilename[0])  , optarg );
              printf("The input file is: %s\n", &(inputfilename[0]));
              break;
          case 'd':
              gfDEVICE=atoi(optarg);

               break;
          case 'f':
                 strcpy ( &(gaugefilename[0])  , optarg );
                 strcpy ( &(fixedgaugename[0])  , "landau_" );
                 strcat ( &(fixedgaugename[0])  , optarg );  
;
              printf("The gauge  file is: %s\n", &(gaugefilename[0]));
              printf("The fixed gauge  file is: %s\n", &(fixedgaugename[0]));
   break;
          case 'h':
          case '?':
          default:
              usage();
              break;
      }
  }
printf("setting device to %d\n", gfDEVICE);
fflush(stdout);
hipSetDevice(gfDEVICE);
int deVice;
hipGetDevice(&deVice);
printf("set device to %d\n", deVice);

  printf("%s\n", hipGetErrorString(hipGetLastError()));


  read_input(&(inputfilename[0]));  
  printf("LX = %d, LY = %d, LZ = %d, T = %d\n", LX, LY, LZ, T);
  
  g_gf = (su3*) malloc(4*VOLUME*sizeof(su3));
  trafo1 = (su3*) malloc(VOLUME*sizeof(su3));
  trafo2 = (su3*) malloc(VOLUME*sizeof(su3));  
     read_gf_ildg(g_gf, &(gaugefilename[0]));

  // if(argc>2){
//    strcpy ( &(gaugefilename[0])  , argv[argc+1] );
//    strcpy ( &(fixedgaugename[0])  , argv[argc+1] );
//    strcat ( &(fixedgaugename[0])  , "_landau" );  
//    printf("The gaugefield file is: %s\n", &(gaugefilename[0]));  
//    read_gf_ildg(g_gf, &(gaugefilename[0]));
//   } 
  
  
 
  printf("Setting random seed to %d\n", randseed);
  PlantSeeds(randseed);


  if(thermflag==1){
   
    if(thermparam.startcond==0){
      unit_init_gauge(g_gf);
    }
    else{
      random_init_gauge(g_gf);
    }
    
    init_thermalization(g_gf);
    init_MT(4*VOLUME/2, 4*4*VOLUME/2); // we need 4 sets of (1/4) (gauss/unif) numbers
                                       // for 4 links per site
    plaq = calc_plaquette(dev_gf,1);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    
    thermalize_gauge();
    
    finalize_thermalization();
  }
  else{
  //unit_init_trafo(trafo1);
  random_init_trafo(trafo1);
  init_gaugefixing(g_gf, trafo1);
  init_MT(VOLUME/2, 4*VOLUME/2); // need one gauss rnd and 4 unif. rnd for all lattice points 
  
  
  //calculate plaquette
  
  
  plaq = calc_plaquette(dev_gf,1);
  printf("%s\n", hipGetErrorString(hipGetLastError()));
  
  F = gauge_functional(g_gf);
  dada = dAdA(g_gf);
  printf("HOST FUNC = %.16e\tHOST dAdA = %.16e\n", F, dada);
  
 
  //small benchmark
  //benchmark();
  //exit(100);
  //end small benchmark

 // do the simulated annealing
 if(saflag==1){
   printf("Starting simulated annealing...\n");
   printf("Tmin = %f, Tmax = %f, N = %d, expo = %f\n", saparam.Tmin, saparam.Tmax, saparam.N, saparam.expo);
   simannealing_gauge(); 
 }


 // do the overrelaxation
 if(orxflag==1){
   printf("Starting overrelaxation...\n");
   ret = overrelax_gauge(orxmaxit, orxeps, orxcheckinterval);  
    if(ret < 0){
     printf("Gauge condition not reached. Aborting...\n");
     finalize_gaugefixing();
     free(trafo1);
     free(trafo2);
     free(g_gf);
     exit(300);
   }
 } 
  
  
   
   #ifdef USETEXTURE
   // apply the trafo dev_gf -> dev_gf2
     /*
     bind_texture_trafo(dev_trafo1);
     dev_apply_trafo<<< griddim, blockdim >>> (dev_gf2, dev_gf, dev_trafo1, dev_nn);
     unbind_texture_trafo();
     */
     
     
     bind_texture_gf(dev_gf);
     plaq = calc_plaquette(dev_gf,1);
     unbind_texture_gf();
   #else
   // apply the trafo dev_gf -> dev_gf  (only one field on GPU)
   
     /* this does not work yet
     dev_apply_trafo<<< griddim, blockdim >>> (dev_gf, dev_gf, dev_trafo1, dev_nn);
     */
     
     plaq = calc_plaquette(dev_gf,1);
   #endif
 
      
  
  printf("%s\n", hipGetErrorString(hipGetLastError()));
  
  // Copy to Host Mem:
  //trafo
  printf("Transferring back to host...\n");
  
  printf("Applying trafo on host...\n");
   
  #ifdef GF_8
   size_t dev_gfsize = 4*VOLUME * sizeof(dev_su3_8);
   hipMemcpy(h2d_trafo, dev_trafo1, dev_gfsize, hipMemcpyDeviceToHost);
   from8tosu3_trafo(trafo1, h2d_trafo); 
  #else
    size_t dev_gfsize = 6*VOLUME * sizeof(dev_su3_2v);
    hipMemcpy(h2d_trafo, dev_trafo1, dev_gfsize, hipMemcpyDeviceToHost);
    from2vtosu3_trafo(trafo1, h2d_trafo);
  #endif
  g_trafo(g_gf, trafo1);
  plaq = mean_plaq(g_gf);
  PLAQ = plaq;
  dada = dAdA(g_gf);
  DADA = dada;
  F = gauge_functional(g_gf);
  FUNC = F;
  printf("Final HOST values:\n");
  printf("PLAQ = %.16f\n", PLAQ); 
  printf("F = %.16e \t dAdA = %.16e\t max(dAdA) = %.16e\n", FUNC, DADA, maxDADA); 

  
  printf("Writing out the gauge fixed field ...");
  ret = write_gf_ildg(g_gf, &(fixedgaugename[0]), 64);
  if(ret!=0){
    fprintf(stderr, "Error writing gauge field. Aborting...\n");
    exit(400);
  }
  printf("done.\n");
  
  //gf
  //dev_gfsize = 6*4*VOLUME * sizeof(dev_su3_2v);
  //hipMemcpy(h2d_gf, dev_gf, dev_gfsize, hipMemcpyDeviceToHost);
  
  

  finalize_gaugefixing();
  }
  
  
  
  free(trafo1);
  free(trafo2);
  free(g_gf);
}





